#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
// #include <cutil.h>
// #include <cutil_inline.h>

#define cutilSafeCall // nop

#include "userapp.cuh"

#define N (8)

int
main(void)
{
    int i, t;
    float a[N], b[N], c[N];

    float *d_a, *d_b, *d_c;
    cutilSafeCall(hipMalloc((void**) &d_a, sizeof(float) * N));
    cutilSafeCall(hipMalloc((void**) &d_b, sizeof(float) * N));
    cutilSafeCall(hipMalloc((void**) &d_c, sizeof(float) * N));

    for (t = 0; t < 3; t++) {
        printf("try %d\n", t);
        for (i = 0; i < N; i++) {
            a[i] = rand()%64;
            b[i] = rand()%64;
        }
        cutilSafeCall(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
        cutilSafeCall(hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));
        int nth = 4;
        dim3 threads(nth, 1, 1);
        dim3 grids((N+nth-1)/nth, 1, 1);
        vecAdd<<<grids, threads>>>(d_a, d_b, d_c);
        cutilSafeCall(hipMemcpy(c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost));    
        for (i = 0; i < N; i++) {
            printf("% 6.2f + % 6.2f = % 7.2f",
                   a[i], b[i], c[i]);
            if (a[i] + b[i] != c[i]) printf("   NG");
            printf("\n");
        }
        printf("\n");
    }

    exit(0);
}
