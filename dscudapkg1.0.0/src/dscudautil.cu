
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hip/driver_types.h>

static int WarnLevel = 2; /* warning message output level. the higher the more verbose.
                             0: no warning (may cause wrong result with g7pkg/scripts/check.csh)
                             1: minimum
                             2: default
                             3: for debugging purpose
                          */
int
dscudaWarnLevel(void)
{
    return WarnLevel;
}

void
dscudaSetWarnLevel(int level)
{
    WarnLevel = level;
}

char *
dscudaMemcpyKindName(hipMemcpyKind kind)
{
    static char *name;

    switch (kind) {
      case hipMemcpyHostToHost:
        name = "hipMemcpyHostToHost";
        break;
      case hipMemcpyHostToDevice:
        name = "hipMemcpyHostToDevice";
        break;
      case hipMemcpyDeviceToHost:
        name = "hipMemcpyDeviceToHost";
        break;
      case hipMemcpyDeviceToDevice:
        name = "hipMemcpyDeviceToDevice";
        break;
      default:
        name = "Invalid hipMemcpyKind";
    }
    return name;
}


const char *
dscudaGetIpaddrString(unsigned int addr)
{
    static char buf[128];
    char *p = (char *)&addr;
    sprintf(buf, "%hhu.%hhu.%hhu.%hhu", p[0], p[1], p[2], p[3]);
    return buf;
}
