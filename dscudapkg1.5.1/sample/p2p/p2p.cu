#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <sys/time.h>

#include <cutil.h>
#include <cutil_inline.h>

static const double MEGA  = 1e6;

enum {
    COPY_BY_MEMCPY,
    COPY_BY_MEMCPYPEER,
};

// warn CUDA API errors, but do not exit.
#define unsafeCall(err)           __unsafeCall   (err, __FILE__, __LINE__)

static inline void
__unsafeCall(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err) {
        fprintf(stderr, "%s(%i) : __unsafeCall() Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
    }
}

static void
get_cputime(double *nowp, double *deltap)
{
    struct timeval t;
    double now0;

    gettimeofday(&t, NULL);
    now0 = t.tv_sec + t.tv_usec/1000000.0;
    *deltap = now0 - *nowp;
    *nowp   = now0;
}

int
main(int argc, char **argv)
{
    double ratio = 2.5;
    double nloop = 2e8;
    double sized;
    double now, dt;
    int maxsize = 1024 * 1024 * 10.0;
    int i, j, size;
    int *bufA, *bufB;
    int *dbufA, *dbufB;
    int srcdev, dstdev;
    int copyapi;
    char copyapistr[128];

    if (argc < 4) {
        fprintf(stderr,
                "copies an array of int from one device to another.\n"
                "usage: %s <c|p> <src_dev> <dst_dev>\n"
                "           'c' for hipMemcpy()\n"
                "           'p' for hipMemcpyPeer()\n",
                argv[0]);
        exit(1);
    }
    switch (argv[1][0]) {
      case 'c':
        copyapi = COPY_BY_MEMCPY;
        sprintf(copyapistr, "hipMemcpy() ");
        break;
      case 'p':
        copyapi = COPY_BY_MEMCPYPEER;
        sprintf(copyapistr, "hipMemcpyPeer() ");
        break;
      default:
        fprintf(stderr, "arg1 should be 'c' or 'p'.\n");
        exit(1);
    }

    srcdev = atoi(argv[2]);
    dstdev = atoi(argv[3]);
    fprintf(stderr, "%s from device %d to device %d.\n",
            copyapistr, srcdev, dstdev);

    bufA = (int *)malloc(maxsize);
    bufB = (int *)malloc(maxsize);

    cutilSafeCall(hipSetDevice(srcdev));
    hipMalloc((void**) &dbufA, maxsize);
    cutilSafeCall(hipSetDevice(dstdev));
    hipMalloc((void**) &dbufB, maxsize);

    // set randomly generated data to the source device.
    for (i = 0; i < maxsize / sizeof(int); i++) {
        bufA[i] = rand() % 64;
        bufB[i] = rand() % 64;
    }
    cutilSafeCall(hipSetDevice(srcdev));
    cutilSafeCall(hipMemcpy(dbufA, bufA, maxsize, hipMemcpyHostToDevice));
    cutilSafeCall(hipSetDevice(dstdev));

    // copy data from the source device to the destination device.
    if (copyapi == COPY_BY_MEMCPY) {
        cutilSafeCall(hipMemcpy(dbufB, dbufA, maxsize, hipMemcpyDefault));
    }
    else {
        unsafeCall(hipMemcpyPeer(dbufB, dstdev, dbufA, srcdev, maxsize));
    }

    // send the data back from the destination device to the host.
    cutilSafeCall(hipMemcpy(bufB, dbufB, maxsize, hipMemcpyDeviceToHost));

    // comparing the result with the original to check the correctness of
    // hipMemcpy() / hipMemcpyPeer()
    for (i = 0; i < maxsize / sizeof(int); i++) {
        if (bufA[i] != bufB[i]) {
            fprintf(stderr, "NG\n");
            fprintf(stderr, "bufA[%d]:0x%08x\n", i, bufA[i]);
            fprintf(stderr, "bufB[%d]:0x%08x\n", i, bufB[i]);
            exit(1);
        }
    }
    fprintf(stderr, "OK\n");

    // measure the bandwidth.
    for (sized = 4096; sized < maxsize; sized *= ratio) {
        //    for ( nloop = 2e8, sized = 4096 * 1; ; ) { // !!!
        size = (size_t)sized;

	get_cputime(&now, &dt);
	for (j = 0; j < nloop/size; j++) {

            if (copyapi == COPY_BY_MEMCPY) {
                hipMemcpy(dbufB, dbufA, size, hipMemcpyDefault);
            }
            else {
                hipMemcpyPeer(dbufB, dstdev, dbufA, srcdev, size);
            }
            //            fprintf(stderr, "j:%d\n", j);
        }
        hipDeviceSynchronize();
	get_cputime(&now, &dt);
	printf("%d byte    %f sec    %f MB/s\n",
               size, dt, nloop/MEGA/dt);
	fflush(stdout);
    }

    exit(0);
}
