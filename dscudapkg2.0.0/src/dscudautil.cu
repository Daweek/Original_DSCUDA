
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <sys/time.h>
#include <hip/driver_types.h>

static int WarnLevel = 2; /* warning message output level. the higher the more verbose.
                             0: no warning (may cause wrong result with g7pkg/scripts/check.csh)
                             1: minimum
                             2: default
                             3: for debugging purpose
                          */
int
dscudaWarnLevel(void)
{
    return WarnLevel;
}

void
dscudaSetWarnLevel(int level)
{
    WarnLevel = level;
}

char *
dscudaMemcpyKindName(hipMemcpyKind kind)
{
    static char *name;

    switch (kind) {
      case hipMemcpyHostToHost:
        name = "hipMemcpyHostToHost";
        break;
      case hipMemcpyHostToDevice:
        name = "hipMemcpyHostToDevice";
        break;
      case hipMemcpyDeviceToHost:
        name = "hipMemcpyDeviceToHost";
        break;
      case hipMemcpyDeviceToDevice:
        name = "hipMemcpyDeviceToDevice";
        break;
      case hipMemcpyDefault:
        name = "hipMemcpyDefault";
        break;
      default:
        name = "Invalid hipMemcpyKind";
    }
    return name;
}

#if 0
const char *
dscudaGetIpaddrString(unsigned int addr)
{
    static char buf[128];
    char *p = (char *)&addr;
    sprintf(buf, "%hhu.%hhu.%hhu.%hhu", p[0], p[1], p[2], p[3]);
    return buf;
}
#endif

unsigned int
dscudaServerNameToDevid(char *svrname)
{
    char buf[256];
    char *token;

    strncpy(buf, svrname, sizeof(buf));
    token = strtok(buf, ":");
    token = strtok(NULL, ":");
    return token ? atoi(token) : 0;
}

unsigned int
dscudaServerIpStrToAddr(char *ipstr)
{
    unsigned int addr;
    struct hostent *host;
    host = gethostbyname(ipstr);
    if (!host) {
        fprintf(stderr, "unknown host name: %s\n", ipstr);
        exit(1);
    }
    addr = *(unsigned int *)host->h_addr_list[0];

    return addr;
}

unsigned int
dscudaServerNameToAddr(char *svrname)
{
    char buf[256];
    char *token;

    strncpy(buf, svrname, sizeof(buf));
    token = strtok(buf, ":");
    return dscudaServerIpStrToAddr(token);
}

char *
dscudaAddrToServerIpStr(unsigned int addr)
{
    struct in_addr ia;

    ia.s_addr = addr;
    return inet_ntoa(ia);
}

/*
 *
 * t0 : time of day (in second) the last time this function is called.
 * returns the number of seconds passed since *t0.
 */
double
RCgetCputime(double *t0)
{
    struct timeval t;
    double tnow, dt;

    gettimeofday(&t, NULL);
    tnow = t.tv_sec + t.tv_usec/1000000.0;
    dt = tnow - *t0;
    *t0 = tnow;
    return dt;
}


int
dscudaAlignUp(int off, int align)
{
    return ((off) + (align) - 1) & ~((align) - 1);
}

unsigned int
dscudaRoundUp(unsigned int src, unsigned int by)
{
    unsigned int dst = ((src - 1) / by + 1) * by;
    return dst;
}

