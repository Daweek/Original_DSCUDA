#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime_api.h>

#define cutilSafeCall checkCudaErrors
#define N (8)

namespace foo {

__constant__ float MyVar0;

__global__ void
vecAdd(float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i] + MyVar0;
}

__global__ void
vecMul(float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

template <typename T0, typename T1> __global__ void
vecAddT(T1 *a, T1 *b, T0 *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

static int
local_func(void)
{
    int i, t;
    float floatvar;
    float a[N], b[N], c[N];
    float *d_a, *d_b, *d_c;
    double *dd_a, *dd_b, *dd_c;
    void (*func)(float *, float *, float *);
    float coeff;

    cutilSafeCall(hipMalloc((void**) &d_a, sizeof(float) * N));
    cutilSafeCall(hipMalloc((void**) &d_b, sizeof(float) * N));
    cutilSafeCall(hipMalloc((void**) &d_c, sizeof(float) * N));

    for (t = 0; t < 3; t++) {
        printf("try %d\n", t);
        for (i = 0; i < N; i++) {
            a[i] = rand()%64;
            b[i] = rand()%64;
            coeff = rand()%64;
        }
        cutilSafeCall(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
        cutilSafeCall(hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

        //        fprintf(stderr, "MyVar0: 0x%016llx  &::MyVar0: 0x%016llx\n", MyVar0, &::MyVar0);

        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(MyVar0), &coeff, sizeof(float), 0, hipMemcpyHostToDevice));
        cutilSafeCall(hipMemcpyFromSymbol(&floatvar, HIP_SYMBOL(MyVar0), sizeof(float), 0, hipMemcpyDeviceToHost));
        printf("floatvar:%f\n", floatvar);

        int nth = 4;
        dim3 threads(nth, 1, 1);
        dim3 grids((N+nth-1)/nth, 1, 1);
        vecAdd<<<grids, threads>>>(d_a, d_b, d_c);
        //        vecAddT<float, float><<<grids, threads>>>(d_a, d_b, d_c);
        //        vecAddT<float, double><<<grids, threads>>>(dd_a, dd_b, d_c);

        func = vecAdd;
        //        fprintf(stderr, ">>>>%x\n", func);
        cutilSafeCall(hipMemcpy(c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost));
        for (i = 0; i < N; i++) {
            printf("% 6.2f + % 6.2f + % 6.2f = % 7.2f",
                   a[i], b[i], coeff, c[i]);
            if (a[i] + b[i] + coeff != c[i]) printf("   NG");
            printf("\n");
        }
        printf("\n");
    }

    exit(0);
}
}

#ifdef NOMAIN
int _unused_main(void)
#else
int main(void)
#endif
{
    foo::local_func();
    exit(0);
}
