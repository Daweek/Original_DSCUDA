extern void dscuda_prog_1(struct svc_req *rqstp, register SVCXPRT *transp);

static int rpcUnpackKernelParam(hipFunction_t *kfuncp, RCargs *argsp);
static void setupRpc(void);

static int
rpcUnpackKernelParam(hipFunction_t *kfuncp, RCargs *argsp)
{
    hipError_t cuerr;
    hipFunction_t kfunc = *kfuncp;
    int ival;
    float fval;
    void *pval;
    RCarg noarg;
    RCarg *argp = &noarg;

    noarg.offset = 0;
    noarg.size = 0;

    for (int i = 0; i < argsp->RCargs_len; i++) {
        argp = &(argsp->RCargs_val[i]);

        switch (argp->val.type) {
          case dscudaArgTypeP:
            pval = (void*)&(argp->val.RCargVal_u.address);
            cuerr = cuParamSetv(kfunc, argp->offset, pval, argp->size);
            if (cuerr != hipSuccess) {
                WARN(0, "cuParamSetv(0x%08llx, %d, 0x%08llx, %d) failed. %s\n",
                     kfunc, argp->offset, pval, argp->size,
                     hipGetErrorString((hipError_t)cuerr));
                fatal_error(1);
            }
            break;

          case dscudaArgTypeI:
            ival = argp->val.RCargVal_u.valuei;
            cuerr = cuParamSeti(kfunc, argp->offset, ival);
            if (cuerr != hipSuccess) {
                WARN(0, "cuParamSeti(0x%08llx, %d, %d) failed. %s\n",
                     kfunc, argp->offset, ival,
                     hipGetErrorString((hipError_t)cuerr));
                fatal_error(1);
            }
            break;

          case dscudaArgTypeF:
            fval = argp->val.RCargVal_u.valuef;
            cuerr = cuParamSetf(kfunc, argp->offset, fval);
            if (cuerr != hipSuccess) {
                WARN(0, "cuParamSetv(0x%08llx, %d, %f) failed. %s\n",
                     kfunc, argp->offset, fval,
                     hipGetErrorString((hipError_t)cuerr));
                fatal_error(1);
            }
            break;

          case dscudaArgTypeV:
            pval = argp->val.RCargVal_u.valuev;
            cuerr = cuParamSetv(kfunc, argp->offset, pval, argp->size);
            if (cuerr != hipSuccess) {
                WARN(0, "cuParamSetv(0x%08llx, %d, 0x%08llx, %d) failed. %s\n",
                     kfunc, argp->offset, pval, argp->size,
                     hipGetErrorString((hipError_t)cuerr));
                fatal_error(1);
            }
            break;

          default:
            WARN(0, "rpcUnpackKernelParam: invalid RCargType\n", argp->val.type);
            fatal_error(1);
        }
    }
    return argp->offset + argp->size;
}

static void
setupRpc(void)
{
    register SVCXPRT *transp;
    unsigned long int prog;

    prog = DSCUDA_PROG + ServerId;
    pmap_unset (prog, DSCUDA_VER);

#if 1 // TCP
    transp = svctcp_create(RPC_ANYSOCK, RC_BUFSIZE, RC_BUFSIZE);
    if (transp == NULL) {
        fprintf (stderr, "%s", "cannot create tcp service.");
        exit(1);
    }
    if (!svc_register(transp, prog, DSCUDA_VER, dscuda_prog_1, IPPROTO_TCP)) {
        fprintf (stderr, "unable to register (prog:0x%x DSCUDA_VER:%d, TCP).\n",
        prog, DSCUDA_VER);
        exit(1);
    }

#else // UDP

    transp = svcudp_create(RPC_ANYSOCK);
    if (transp == NULL) {
        fprintf (stderr, "%s", "cannot create udp service.");
        exit(1);
    }
    if (!svc_register(transp, prog, DSCUDA_VER, dscuda_prog_1, IPPROTO_UDP)) {
        fprintf (stderr, "%s", "unable to register (prog, DSCUDA_VER, udp).");
        exit(1);
    }

#endif
}

/*
 * CUDA API stubs
 */

/*
 * Thread Management
 */

dscudaResult *
dscudathreadexitid_1_svc(struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(3, "hipDeviceReset(\n");
    if (!rcuContext) createRcuContext();

    err = hipDeviceReset();
    check_cuda_error(err);
    res.err = err;
    WARN(3, ") done.\n");
    return &res;
}

dscudaResult *
dscudathreadsynchronizeid_1_svc(struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(3, "hipDeviceSynchronize(");
    if (!rcuContext) createRcuContext();

    err = hipDeviceSynchronize();
    check_cuda_error(err);
    res.err = err;
    WARN(3, ") done.\n");
    return &res;
}

dscudaResult *
dscudathreadsetlimitid_1_svc(int limit, RCsize value, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(3, "cudaThreadSetLimit(");
    if (!rcuContext) createRcuContext();

    err = cudaThreadSetLimit((enum hipLimit_t)limit, value);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "%d, %d) done.\n", limit, value);
    return &res;
}

dscudaThreadGetLimitResult *
dscudathreadgetlimitid_1_svc(int limit, struct svc_req *sr)
{
    hipError_t err;
    static dscudaThreadGetLimitResult res;
    size_t value;

    WARN(3, "cudaThreadGetLimit(");
    if (!rcuContext) createRcuContext();

    err = cudaThreadGetLimit(&value, (enum hipLimit_t)limit);
    check_cuda_error(err);
    res.err = err;
    res.value = value;
    WARN(3, "0x%08llx, %d) done.  value:%d\n", &value, limit, value);
    return &res;
}

dscudaResult *
dscudathreadsetcacheconfigid_1_svc(int cacheConfig, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(3, "hipDeviceSetCacheConfig(");
    if (!rcuContext) createRcuContext();

    err = hipDeviceSetCacheConfig((enum hipFuncCache_t)cacheConfig);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "%d) done.\n", cacheConfig);
    return &res;
}

dscudaThreadGetCacheConfigResult *
dscudathreadgetcacheconfigid_1_svc(struct svc_req *sr)
{
    hipError_t err;
    static dscudaThreadGetCacheConfigResult res;
    int cacheConfig;

    WARN(3, "hipDeviceGetCacheConfig(");
    if (!rcuContext) createRcuContext();

    err = hipDeviceGetCacheConfig((enum hipFuncCache_t *)&cacheConfig);
    check_cuda_error(err);
    res.err = err;
    res.cacheConfig = cacheConfig;
    WARN(3, "0x%08llx) done.  cacheConfig:%d\n", &cacheConfig, cacheConfig);
    return &res;
}


/*
 * Error Handling
 */

dscudaResult *
dscudagetlasterrorid_1_svc(struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(5, "hipGetLastError(");
    if (!rcuContext) createRcuContext();

    err = hipGetLastError();
    check_cuda_error(err);
    res.err = err;
    WARN(5, ") done.\n");
    return &res;
}

dscudaResult *
dscudapeekatlasterrorid_1_svc(struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(5, "hipPeekAtLastError(");
    if (!rcuContext) createRcuContext();

    err = hipPeekAtLastError();
    check_cuda_error(err);
    res.err = err;
    WARN(5, ") done.\n");
    return &res;
}

dscudaGetErrorStringResult *
dscudageterrorstringid_1_svc(int err, struct svc_req *sr)
{
    static dscudaGetErrorStringResult res;

    WARN(3, "hipGetErrorString(");
    if (!rcuContext) createRcuContext();

    res.errmsg = (char *)hipGetErrorString((hipError_t)err);
    WARN(3, "%d) done.\n", err);
    return &res;
}


/*
 * Device Management
 */

dscudaGetDeviceResult *
dscudagetdeviceid_1_svc(struct svc_req *sr)
{
    hipError_t err;
    int device;
    static dscudaGetDeviceResult res;

    WARN(3, "hipGetDevice(");
    if (!rcuContext) createRcuContext();

    err = hipGetDevice(&device);
    check_cuda_error(err);
    res.device = Devid2Vdevid[device];
    res.err = err;
    WARN(3, "0x%08llx) done. device:%d  virtual device:%d\n",
         (unsigned long)&device, device, res.device);
    return &res;
}

dscudaGetDeviceCountResult *
dscudagetdevicecountid_1_svc(struct svc_req *sr)
{
    int count;
    static dscudaGetDeviceCountResult res;

    WARN(3, "hipGetDeviceCount(");

#if 0
// this returns # of devices in the system, even if the number of
// valid devices set by hipSetValidDevices() is smaller.
    hipError_t err;
    err = hipGetDeviceCount(&count);
    check_cuda_error(err);
    res.count = count;
    res.err = err;
#else
    res.count = count = Ndevice;
    res.err = hipSuccess;
#endif
    WARN(3, "0x%08llx) done. count:%d\n", (unsigned long)&count, count);
    return &res;
}

dscudaGetDevicePropertiesResult *
dscudagetdevicepropertiesid_1_svc(int device, struct svc_req *sr)
{
    hipError_t err;
    static int firstcall = 1;
    static dscudaGetDevicePropertiesResult res;

    WARN(3, "hipGetDeviceProperties(");

    if (firstcall) {
        firstcall = 0;
        res.prop.RCbuf_val = (char*)malloc(sizeof(hipDeviceProp_t));
        res.prop.RCbuf_len = sizeof(hipDeviceProp_t);
    }
    if (1 < Ndevice) {
        WARN(0, "dscudagetdevicepropertiesid_1_svc() cannot handle multiple devices for now. Ndevice:%d\n",
             Ndevice);
        exit(1);
    }
    err = hipGetDeviceProperties((hipDeviceProp_t *)res.prop.RCbuf_val, Devid[0]);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d) done.\n", (unsigned long)res.prop.RCbuf_val, Devid[0]);
    return &res;
}

dscudaDriverGetVersionResult *
dscudadrivergetversionid_1_svc(struct svc_req *sr)
{
    hipError_t err;
    int ver;
    static dscudaDriverGetVersionResult res;

    WARN(3, "hipDriverGetVersion(");

    if (!rcuContext) createRcuContext();

    err = hipDriverGetVersion(&ver);
    check_cuda_error(err);
    res.ver = ver;
    res.err = err;
    WARN(3, "0x%08llx) done.\n", (unsigned long)&ver);
    return &res;
}

dscudaRuntimeGetVersionResult *
dscudaruntimegetversionid_1_svc(struct svc_req *sr)
{
    hipError_t err;
    int ver;
    static dscudaRuntimeGetVersionResult res;

    WARN(3, "hipRuntimeGetVersion(");

    if (!rcuContext) createRcuContext();

    err = hipRuntimeGetVersion(&ver);
    check_cuda_error(err);
    res.ver = ver;
    res.err = err;
    WARN(3, "0x%08llx) done.\n", (unsigned long)&ver);
    return &res;
}

dscudaResult *
dscudasetdeviceid_1_svc(int device, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(3, "hipSetDevice(");

    if (rcuContext) destroyRcuContext();

    rcuDevice = Devid[device];
    err = createRcuContext();
    res.err = err;
    WARN(3, "%d) done.  rcuDevice: %d\n",
         device, rcuDevice);
    return &res;
}

dscudaResult *
dscudasetdeviceflagsid_1_svc(unsigned int flags, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(3, "hipSetDeviceFlags(");

    /* hipSetDeviceFlags() API should be called only when
     * the device is not active, i.e., rcuContext does not exist.
     * Before invoking the API, destroy the context if valid. */
    if (rcuContext) destroyRcuContext();

    err = hipSetDeviceFlags(flags);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08x)\n", flags);

    return &res;
}

dscudaChooseDeviceResult *
dscudachoosedeviceid_1_svc(RCbuf prop, struct svc_req *sr)
{
    hipError_t err;
    int device;
    static dscudaChooseDeviceResult res;

    WARN(3, "hipGetDevice(");
    if (!rcuContext) createRcuContext();

    err = hipChooseDevice(&device, (const struct hipDeviceProp_t *)&prop.RCbuf_val);
    check_cuda_error(err);
    res.device = Devid2Vdevid[device];
    res.err = err;
    WARN(3, "0x%08llx) done. device:%d  virtual device:%d\n",
         (unsigned long)&device, device, res.device);
    return &res;
}


dscudaResult *
dscudadevicesynchronize_1_svc(struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;

    WARN(3, "hipDeviceSynchronize(");
    if (!rcuContext) createRcuContext();

    err = hipDeviceSynchronize();
    check_cuda_error(err);
    res.err = err;
    WARN(3, ") done.\n");

    return &res;
}

dscudaResult *
dscudadevicereset_1_svc(struct svc_req *sr)
{
    hipError_t err;
    bool all = true;
    static dscudaResult res;

    WARN(3, "hipDeviceReset(");
    if (!rcuContext) createRcuContext();

    err = hipDeviceReset();
    check_cuda_error(err);
    res.err = err;
    releaseModules(all);
    WARN(3, ") done.\n");

    return &res;
}

/*
 * Stream Management
 */

dscudaStreamCreateResult *
dscudastreamcreateid_1_svc(struct svc_req *sr)
{
    static dscudaStreamCreateResult res;
    hipError_t err;
    hipStream_t stream;

    WARN(3, "hipStreamCreate(");
    if (!rcuContext) createRcuContext();
    err = hipStreamCreate(&stream);
    res.stream = (RCadr)stream;
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done. stream:0x%08llx\n", &stream, stream);

    return &res;
}

dscudaResult *
dscudastreamdestroyid_1_svc(RCstream stream, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipStreamDestroy(");
    if (!rcuContext) createRcuContext();
    err = hipStreamDestroy((hipStream_t)stream);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", stream);

    return &res;
}

dscudaResult *
dscudastreamwaiteventid_1_svc(RCstream stream, RCevent event, unsigned int flags, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipStreamWaitEvent(");
    if (!rcuContext) createRcuContext();
    err = hipStreamWaitEvent((hipStream_t)stream, (hipEvent_t)event, flags);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx 0x%08llx, 0x%08x) done.\n", stream, event, flags);

    return &res;
}

dscudaResult *
dscudastreamsynchronizeid_1_svc(RCstream stream, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipStreamSynchronize(");
    if (!rcuContext) createRcuContext();
    err = hipStreamSynchronize((hipStream_t)stream);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", stream);

    return &res;
}

dscudaResult *
dscudastreamqueryid_1_svc(RCstream stream, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipStreamQuery(");
    if (!rcuContext) createRcuContext();
    err = hipStreamQuery((hipStream_t)stream);
    // should not check error due to the nature of this API.
    // check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", stream);

    return &res;
}

/*
 * Event Management
 */

dscudaEventCreateResult *
dscudaeventcreateid_1_svc(struct svc_req *sr)
{
    static dscudaEventCreateResult res;
    hipError_t err;
    hipEvent_t event;

    WARN(3, "hipEventCreate(");
    if (!rcuContext) createRcuContext();
    err = hipEventCreate(&event);
    res.event = (RCadr)event;
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done. event:0x%08llx\n", &event, event);

    return &res;
}

dscudaEventCreateResult *
dscudaeventcreatewithflagsid_1_svc(unsigned int flags, struct svc_req *sr)
{
    static dscudaEventCreateResult res;
    hipError_t err;
    hipEvent_t event;

    WARN(3, "hipEventCreateWithFlags(");
    if (!rcuContext) createRcuContext();
    err = hipEventCreateWithFlags(&event, flags);
    res.event = (RCadr)event;
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, 0x%08x) done. event:0x%08llx\n", &event, flags, event);

    return &res;
}

dscudaResult *
dscudaeventdestroyid_1_svc(RCevent event, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipEventDestroy(");
    if (!rcuContext) createRcuContext();
    err = hipEventDestroy((hipEvent_t)event);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", event);

    return &res;
}

dscudaEventElapsedTimeResult *
dscudaeventelapsedtimeid_1_svc(RCevent start, RCevent end, struct svc_req *sr)
{
    static dscudaEventElapsedTimeResult res;
    hipError_t err;
    float millisecond;

    WARN(3, "hipEventElapsedTime(");
    if (!rcuContext) createRcuContext();
    err = hipEventElapsedTime(&millisecond, (hipEvent_t)start, (hipEvent_t)end);
    check_cuda_error(err);
    res.ms = millisecond;
    res.err = err;
    WARN(3, "%5.3f 0x%08llx 0x%08llx) done.\n", millisecond, start, end);

    return &res;
}

dscudaResult *
dscudaeventrecordid_1_svc(RCevent event, RCstream stream, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipEventRecord(");
    if (!rcuContext) createRcuContext();
    err = hipEventRecord((hipEvent_t)event, (hipStream_t)stream);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx 0x%08llx) done.\n", event, stream);

    return &res;
}

dscudaResult *
dscudaeventsynchronizeid_1_svc(RCevent event, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipEventSynchronize(");
    if (!rcuContext) createRcuContext();
    err = hipEventSynchronize((hipEvent_t)event);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", event);

    return &res;
}

dscudaResult *
dscudaeventqueryid_1_svc(RCevent event, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipEventQuery(");
    if (!rcuContext) createRcuContext();
    err = hipEventQuery((hipEvent_t)event);
    // should not check error due to the nature of this API.
    // check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", event);

    return &res;
}


dscudaFuncGetAttributesResult *
dscudafuncgetattributesid_1_svc(int moduleid, char *kname, struct svc_req *sr)
{
    static dscudaFuncGetAttributesResult res;
    hipError_t err;
    hipFunction_t kfunc;

    if (!rcuContext) createRcuContext();

    err = getFunctionByName(&kfunc, kname, moduleid);
    check_cuda_error((hipError_t)err);

    WARN(3, "hipFuncGetAttribute(");
    err = hipFuncGetAttribute(&res.attr.binaryVersion, HIP_FUNC_ATTRIBUTE_BINARY_VERSION, kfunc);
    check_cuda_error((hipError_t)err);
    WARN(3, "0x%08llx, %d, 0x%08llx) done.\n", &res.attr.binaryVersion, HIP_FUNC_ATTRIBUTE_BINARY_VERSION, kfunc);

    err = hipFuncGetAttribute((int *)&res.attr.constSizeBytes, HIP_FUNC_ATTRIBUTE_CONST_SIZE_BYTES, kfunc);
    check_cuda_error((hipError_t)err);
    WARN(3, "0x%08llx, %d, 0x%08llx) done.\n", &res.attr.constSizeBytes, HIP_FUNC_ATTRIBUTE_CONST_SIZE_BYTES, kfunc);

    err = hipFuncGetAttribute((int *)&res.attr.localSizeBytes, HIP_FUNC_ATTRIBUTE_LOCAL_SIZE_BYTES, kfunc);
    check_cuda_error((hipError_t)err);
    WARN(3, "0x%08llx, %d, 0x%08llx) done.\n", &res.attr.localSizeBytes, HIP_FUNC_ATTRIBUTE_LOCAL_SIZE_BYTES, kfunc);

    err = hipFuncGetAttribute(&res.attr.maxThreadsPerBlock, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, kfunc);
    check_cuda_error((hipError_t)err);
    WARN(3, "0x%08llx, %d, 0x%08llx) done.\n", &res.attr.maxThreadsPerBlock, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, kfunc);

    err = hipFuncGetAttribute(&res.attr.numRegs, HIP_FUNC_ATTRIBUTE_NUM_REGS, kfunc);
    check_cuda_error((hipError_t)err);
    WARN(3, "0x%08llx, %d, 0x%08llx) done.\n", &res.attr.numRegs, HIP_FUNC_ATTRIBUTE_NUM_REGS, kfunc);

    err = hipFuncGetAttribute(&res.attr.ptxVersion, HIP_FUNC_ATTRIBUTE_PTX_VERSION, kfunc);
    check_cuda_error((hipError_t)err);
    WARN(3, "0x%08llx, %d, 0x%08llx) done.\n", &res.attr.ptxVersion, HIP_FUNC_ATTRIBUTE_PTX_VERSION, kfunc);

    err = hipFuncGetAttribute((int *)&res.attr.sharedSizeBytes, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, kfunc);
    check_cuda_error((hipError_t)err);
    WARN(3, "0x%08llx, %d, 0x%08llx) done.\n", &res.attr.sharedSizeBytes, HIP_FUNC_ATTRIBUTE_SHARED_SIZE_BYTES, kfunc);

    res.err = err;

    return &res;
}

/*
 * Memory Management
 */

dscudaMallocResult * 
dscudamallocid_1_svc(RCsize size, struct svc_req *sr)
{
    static dscudaMallocResult res;
    hipError_t err;
    int *devadr;

    WARN(3, "hipMalloc(");
    if (!rcuContext) createRcuContext();
    err = hipMalloc((void**)&devadr, size);
    res.devAdr = (RCadr)devadr;
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d) done. devadr:0x%08llx\n", &devadr, size, devadr);

    return &res;
}

dscudaResult *
dscudafreeid_1_svc(RCadr mem, struct svc_req *)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipFree(");
    if (!rcuContext) createRcuContext();
    err = hipFree((void*)mem);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", mem);

    return &res;
}

dscudaMemcpyH2HResult *
dscudamemcpyh2hid_1_svc(RCadr dst, RCbuf srcbuf, RCsize count, struct svc_req *sr)
{
    static dscudaMemcpyH2HResult res;
    WARN(0, "dscudaMemcpy() does not support hipMemcpyHostToHost transfer yet.\n");
    return &res;
}

dscudaResult *
dscudamemcpyh2did_1_svc(RCadr dst, RCbuf srcbuf, RCsize count, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipMemcpy(");
    if (!rcuContext) createRcuContext();
    err = hipMemcpy((void*)dst, srcbuf.RCbuf_val, count, hipMemcpyHostToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, 0x%08lx, %d, %s) done.\n",
            dst, (unsigned long)srcbuf.RCbuf_val, count, dscudaMemcpyKindName(hipMemcpyHostToDevice));
    return &res;
}

dscudaMemcpyD2HResult *
dscudamemcpyd2hid_1_svc(RCadr src, RCsize count, struct svc_req *sr)
{
    static RCsize maxcount = 0;
    static dscudaMemcpyD2HResult res;
    hipError_t err;

    WARN(3, "hipMemcpy(");
    if (!rcuContext) createRcuContext();
    if (maxcount == 0) {
        res.buf.RCbuf_val = NULL;
    }
    if (maxcount < count) {
        res.buf.RCbuf_val = (char*)realloc(res.buf.RCbuf_val, count);
        maxcount = count;
    }
    res.buf.RCbuf_len = count;
    err = hipMemcpy(res.buf.RCbuf_val, (const void*)src, count, hipMemcpyDeviceToHost);
    WARN(3, "0x%08lx, 0x%08llx, %d, %s) done.\n",
         (unsigned long)res.buf.RCbuf_val, src, count, dscudaMemcpyKindName(hipMemcpyDeviceToHost));
    check_cuda_error(err);
    res.err = err;

#if 0 // destroy some part of the returning data. debugging purpose only.
    {
        srand48(time(NULL));
        if (ServerId == 0 && drand48() < 1.0/100.0) {
            res.buf.RCbuf_val[0] = 0;
        }
    }
#endif

    return &res;
}

dscudaResult *
dscudamemcpyd2did_1_svc(RCadr dst, RCadr src, RCsize count, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;
    WARN(3, "hipMemcpy(");
    err = hipMemcpy((void *)dst, (void *)src, count, hipMemcpyDeviceToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, 0x%08llx, %d, %s) done.\n",
            dst, src, count, dscudaMemcpyKindName(hipMemcpyDeviceToDevice));
    return &res;
}

dscudaMallocArrayResult *
dscudamallocarrayid_1_svc(RCchanneldesc desc, RCsize width, RCsize height, unsigned int flags, struct svc_req *sr)
{
    static dscudaMallocArrayResult res;
    hipError_t err;
    hipArray *devadr;
    hipChannelFormatDesc descbuf = hipCreateChannelDesc(desc.x, desc.y, desc.z, desc.w, (enum hipChannelFormatKind)desc.f);

    WARN(3, "hipMallocArray(");
    if (!rcuContext) createRcuContext();
    err = hipMallocArray((hipArray**)&devadr, &descbuf, width, height, flags);
    res.array = (RCadr)devadr;
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, 0x%08llx, %d, %d, 0x%08x) done. devadr:0x%08llx\n",
         &devadr, &descbuf, width, height, flags, devadr)

    return &res;
}

dscudaResult *
dscudafreearrayid_1_svc(RCadr array, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipFreeArray(");
    if (!rcuContext) createRcuContext();
    err = hipFreeArray((hipArray*)array);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", array);

    return &res;
}

dscudaMemcpyToArrayH2HResult *
dscudamemcpytoarrayh2hid_1_svc(RCadr dst, RCsize wOffset, RCsize hOffset, RCbuf src, RCsize count, struct svc_req *sr)
{
    static dscudaMemcpyToArrayH2HResult res;
    WARN(0, "dscudaMemcpyToArray() does not support hipMemcpyHostToHost transfer yet.\n");
    return &res;
}

dscudaResult *
dscudamemcpytoarrayh2did_1_svc(RCadr dst, RCsize wOffset, RCsize hOffset, RCbuf src, RCsize count, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipMemcpyToArray(");
    if (!rcuContext) createRcuContext();
    err = hipMemcpyToArray((hipArray *)dst, wOffset, hOffset, src.RCbuf_val, count, hipMemcpyHostToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, %d, 0x%08llx, %d, %s) done.\n",
         dst, wOffset, hOffset, (unsigned long)src.RCbuf_val, count, dscudaMemcpyKindName(hipMemcpyHostToDevice));
    return &res;
}

dscudaMemcpyToArrayD2HResult *
dscudamemcpytoarrayd2hid_1_svc(RCsize wOffset, RCsize hOffset, RCadr src, RCsize count, struct svc_req *sr)
{
    static dscudaMemcpyToArrayD2HResult res;
    WARN(0, "dscudaMemcpyToArray() does not support hipMemcpyDeviceToHost transfer yet.\n");
    return &res;
}

dscudaResult *
dscudamemcpytoarrayd2did_1_svc(RCadr dst, RCsize wOffset, RCsize hOffset, RCadr src, RCsize count, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;
    WARN(3, "hipMemcpyToArray(");
    err = hipMemcpyToArray((hipArray *)dst, wOffset, hOffset, (void *)src, count, hipMemcpyDeviceToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, %d, 0x%08llx, %d, %s) done.\n",
         dst, wOffset, hOffset, src, count, dscudaMemcpyKindName(hipMemcpyDeviceToDevice));
    return &res;
}

dscudaResult *
dscudamemcpytosymbolh2did_1_svc(int moduleid, char *symbol, RCbuf src, RCsize count, RCsize offset, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;
    hipDeviceptr_t gsptr;
    size_t gssize;

    WARN(3, "hipMemcpyToSymbol(");
    if (!rcuContext) createRcuContext();

    getGlobalSymbol(moduleid, symbol, &gsptr, &gssize);
    err = hipMemcpy((char *)gsptr + offset, src.RCbuf_val, count, hipMemcpyHostToDevice);
                             
    WARN(3, "0x%08lx, 0x%08lx, %d, %d, %s) done. module name:%s  symbol:%s\n",
         gsptr, (unsigned long)src.RCbuf_val, count, offset,
         dscudaMemcpyKindName(hipMemcpyHostToDevice),
         Modulelist[moduleid].name, symbol);
    check_cuda_error(err);
    res.err = err;
    return &res;
}

dscudaResult *
dscudamemcpytosymbold2did_1_svc(int moduleid, char *symbol, RCadr src, RCsize count, RCsize offset, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;
    hipDeviceptr_t gsptr;
    size_t gssize;

    WARN(3, "hipMemcpyToSymbol(");
    if (!rcuContext) createRcuContext();

    getGlobalSymbol(moduleid, symbol, &gsptr, &gssize);

    err = hipMemcpy((char *)gsptr + offset, (void*)src, count, hipMemcpyDeviceToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, 0x%08llx, %d, %d, %s) done.\n",
         gsptr, (unsigned long)src, count, offset,
         dscudaMemcpyKindName(hipMemcpyDeviceToDevice));

    return &res;
}

dscudaMemcpyFromSymbolD2HResult *
dscudamemcpyfromsymbold2hid_1_svc(int moduleid, char *symbol, RCsize count, RCsize offset, struct svc_req *sr)
{
    static RCsize maxcount = 0;
    static dscudaMemcpyFromSymbolD2HResult res;
    hipError_t err;
    hipDeviceptr_t gsptr;
    size_t gssize;

    WARN(3, "hipMemcpyFromSymbol(");
    if (!rcuContext) createRcuContext();
    if (maxcount == 0) {
        res.buf.RCbuf_val = NULL;
    }
    if (maxcount < count) {
        res.buf.RCbuf_val = (char*)realloc(res.buf.RCbuf_val, count);
        maxcount = count;
    }
    res.buf.RCbuf_len = count;

    getGlobalSymbol(moduleid, symbol, &gsptr, &gssize);
    err = hipMemcpy(res.buf.RCbuf_val, (char *)gsptr + offset, count, hipMemcpyDeviceToHost);
                             
    WARN(3, "0x%08llx, 0x%08llx, %d, %d, %s) done. module name:%s  symbol:%s\n",
         (unsigned long)res.buf.RCbuf_val, gsptr, count, offset,
         dscudaMemcpyKindName(hipMemcpyDeviceToHost),
         Modulelist[moduleid].name, symbol);
    check_cuda_error(err);
    res.err = err;
    return &res;
}

dscudaResult *
dscudamemcpyfromsymbold2did_1_svc(int moduleid, RCadr dst, char *symbol, RCsize count, RCsize offset, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;
    hipDeviceptr_t gsptr;
    size_t gssize;

    WARN(3, "hipMemcpyFromSymbol(");
    if (!rcuContext) createRcuContext();

    getGlobalSymbol(moduleid, symbol, &gsptr, &gssize);

    err = hipMemcpy((void*)dst, (char *)gsptr + offset, count, hipMemcpyDeviceToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, 0x%08llx, %d, %d, %s) done.\n",
         (unsigned long)dst, gsptr, count, offset,
         dscudaMemcpyKindName(hipMemcpyDeviceToDevice));

    return &res;
}


dscudaResult *
dscudamemcpytosymbolasynch2did_1_svc(int moduleid, char *symbol, RCbuf src, RCsize count, RCsize offset, RCstream stream, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;
    hipDeviceptr_t gsptr;
    size_t gssize;

    WARN(3, "hipMemcpyToSymbolAsync(");
    if (!rcuContext) createRcuContext();
    getGlobalSymbol(moduleid, symbol, &gsptr, &gssize);
    err = hipMemcpyAsync((char *)gsptr + offset, src.RCbuf_val, count, hipMemcpyHostToDevice, (hipStream_t)stream);
                             
    WARN(3, "0x%08lx, 0x%08lx, %d, %d, %s, 0x%08llx) done. module name:%s  symbol:%s\n",
         gsptr, (unsigned long)src.RCbuf_val, count, offset,
         dscudaMemcpyKindName(hipMemcpyHostToDevice), stream,
         Modulelist[moduleid].name, symbol);

    check_cuda_error(err);
    res.err = err;
    return &res;
}

dscudaResult *
dscudamemcpytosymbolasyncd2did_1_svc(int moduleid, char *symbol, RCadr src, RCsize count, RCsize offset, RCstream stream, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;
    hipDeviceptr_t gsptr;
    size_t gssize;

    WARN(3, "hipMemcpyToSymbolAsync(");
    if (!rcuContext) createRcuContext();

    getGlobalSymbol(moduleid, symbol, &gsptr, &gssize);

    err = hipMemcpyAsync((char *)gsptr + offset, (void*)src, count, hipMemcpyDeviceToDevice, (hipStream_t)stream);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08lx, 0x%08lx, %d, %d, %s, 0x%08llx) done.\n",
         gsptr, (unsigned long)src, count, offset, stream,
         dscudaMemcpyKindName(hipMemcpyDeviceToDevice));

    return &res;
}

dscudaMemcpyFromSymbolAsyncD2HResult *
dscudamemcpyfromsymbolasyncd2hid_1_svc(int moduleid, char *symbol, RCsize count, RCsize offset, RCstream stream, struct svc_req *sr)
{
    static RCsize maxcount = 0;
    static dscudaMemcpyFromSymbolAsyncD2HResult res;
    hipError_t err;
    hipDeviceptr_t gsptr;
    size_t gssize;

    WARN(3, "hipMemcpyFromSymbolAsync(");
    if (!rcuContext) createRcuContext();
    if (maxcount == 0) {
        res.buf.RCbuf_val = NULL;
    }
    if (maxcount < count) {
        res.buf.RCbuf_val = (char*)realloc(res.buf.RCbuf_val, count);
        maxcount = count;
    }
    res.buf.RCbuf_len = count;

    getGlobalSymbol(moduleid, symbol, &gsptr, &gssize);
    err = hipMemcpyAsync(res.buf.RCbuf_val, (char *)gsptr + offset, count, hipMemcpyDeviceToHost, (hipStream_t)stream);
                             
    WARN(3, "0x%08lx, 0x%08lx, %d, %d, %s, 0x%08llx) done. module name:%s  symbol:%s\n",
         (unsigned long)res.buf.RCbuf_val, gsptr, count, offset, stream,
         dscudaMemcpyKindName(hipMemcpyDeviceToHost),
         Modulelist[moduleid].name, symbol);
    check_cuda_error(err);
    res.err = err;
    return &res;
}

dscudaResult *
dscudamemcpyfromsymbolasyncd2did_1_svc(int moduleid, RCadr dst, char *symbol, RCsize count, RCsize offset, RCstream stream, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;
    hipDeviceptr_t gsptr;
    size_t gssize;

    WARN(3, "hipMemcpyFromSymbolAsync(");
    if (!rcuContext) createRcuContext();

    getGlobalSymbol(moduleid, symbol, &gsptr, &gssize);

    err = hipMemcpyAsync((void*)dst, (char *)gsptr + offset, count, hipMemcpyDeviceToDevice, (hipStream_t)stream);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08lx, 0x%08lx, %d, %d, %s, 0x%08llx) done.\n",
         (unsigned long)dst, gsptr, count, offset, stream,
         dscudaMemcpyKindName(hipMemcpyDeviceToDevice));

    return &res;
}


dscudaResult *
dscudamemsetid_1_svc(RCadr dst, int value, RCsize count, struct svc_req *sq)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipMemset(");
    if (!rcuContext) createRcuContext();
    err = hipMemset((void *)dst, value, count);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, %d) done.\n", dst, value, count);
    return &res;
}

dscudaHostAllocResult *
dscudahostallocid_1_svc(RCsize size, unsigned int flags, struct svc_req *sr)
{
    static dscudaHostAllocResult res;
    hipError_t err;
    int *devadr;

    WARN(3, "hipHostAlloc(");
    if (!rcuContext) createRcuContext();
    err = hipHostAlloc((void**)&devadr, size, flags);
    res.pHost = (RCadr)devadr;
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, 0x%08x) done.\n", res.pHost, size, flags);

    return &res;
}

dscudaMallocHostResult *
dscudamallochostid_1_svc(RCsize size, struct svc_req *sr)
{
    static dscudaMallocHostResult res;
    hipError_t err;
    int *devadr;

    WARN(3, "hipHostMalloc(");
    if (!rcuContext) createRcuContext();
    err = hipHostMalloc((void**)&devadr, size);
    res.ptr = (RCadr)devadr;
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d) done. devadr:0x%08llx\n", &devadr, size, devadr);

    return &res;
}

dscudaResult *
dscudafreehostid_1_svc(RCadr ptr, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipHostFree(");
    if (!rcuContext) createRcuContext();
    err = hipHostFree((void*)ptr);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx) done.\n", ptr);

    return &res;
}

dscudaHostGetDevicePointerResult *
dscudahostgetdevicepointerid_1_svc(RCadr pHost, unsigned int flags , struct svc_req *sr)
{
    hipError_t err;
    static dscudaHostGetDevicePointerResult res;
    RCadr pDevice;

    WARN(3, "hipHostGetDevicePointer(");
    if (!rcuContext) createRcuContext();

    err = hipHostGetDevicePointer((void **)&pDevice, (void *)pHost, flags);
    check_cuda_error(err);
    res.pDevice = pDevice;
    res.err = err;
    WARN(3, ") done.\n");
    return &res;
}

dscudaHostGetFlagsResult *
dscudahostgetflagsid_1_svc(RCadr pHost, struct svc_req *sr)
{
    hipError_t err;
    static dscudaHostGetFlagsResult res;
    unsigned int flags;

    WARN(3, "hipHostGetFlags(");
    if (!rcuContext) createRcuContext();

    err = hipHostGetFlags(&flags, (void *)pHost);
    check_cuda_error(err);
    res.err = err;
    res.flags = flags;
    WARN(3, ") done.\n");
    return &res;
}

dscudaMemcpyAsyncH2HResult *
dscudamemcpyasynch2hid_1_svc(RCadr dst, RCbuf src, RCsize count, RCstream stream, struct svc_req *sr)
{
    static dscudaMemcpyAsyncH2HResult res;
    WARN(0, "dscudaMemcpyAsync() does not support hipMemcpyHostToHost transfer yet.\n");
    return &res;
}

dscudaResult *
dscudamemcpyasynch2did_1_svc(RCadr dst, RCbuf src, RCsize count, RCstream stream, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipMemcpyAsync(");
    if (!rcuContext) createRcuContext();
    err = hipMemcpyAsync((void*)dst, src.RCbuf_val, count, hipMemcpyHostToDevice, (hipStream_t)stream);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08lx, 0x%08lx, %d, %s, 0x%08lx) done.\n",
         dst, (unsigned long)src.RCbuf_val, count, dscudaMemcpyKindName(hipMemcpyHostToDevice), stream);
    return &res;
}

dscudaMemcpyAsyncD2HResult *
dscudamemcpyasyncd2hid_1_svc(RCadr src, RCsize count, RCstream stream, struct svc_req *sr)
{
    static RCsize maxcount = 0;
    hipError_t err;
    static dscudaMemcpyAsyncD2HResult res;

    WARN(3, "hipMemcpyAsync(");
    if (!rcuContext) createRcuContext();
    if (maxcount == 0) {
        res.buf.RCbuf_val = NULL;
    }
    if (maxcount < count) {
        res.buf.RCbuf_val = (char*)realloc(res.buf.RCbuf_val, count);
        maxcount = count;
    }
    res.buf.RCbuf_len = count;
    err = hipMemcpyAsync(res.buf.RCbuf_val, (const void*)src, count, hipMemcpyDeviceToHost, (hipStream_t)stream);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, 0x%08llx, %d, %s, 0x%08llx) done.\n",
         (unsigned long)res.buf.RCbuf_val, src, count, dscudaMemcpyKindName(hipMemcpyDeviceToHost), stream);
    return &res;
}

dscudaResult *
dscudamemcpyasyncd2did_1_svc(RCadr dst, RCadr src, RCsize count, RCstream stream, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;
    WARN(3, "hipMemcpyAsync(");
    err = hipMemcpyAsync((void *)dst, (void *)src, count, hipMemcpyDeviceToDevice, (hipStream_t)stream);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, 0x%08llx, %d, %s, 0x%08llx) done.\n",
         dst, src, count, dscudaMemcpyKindName(hipMemcpyDeviceToDevice), stream);
    return &res;
}


dscudaMallocPitchResult *
dscudamallocpitchid_1_svc(RCsize width, RCsize height, struct svc_req *sr)
{
    static dscudaMallocPitchResult res;
    hipError_t err;
    int *devadr;
    size_t pitch;

    WARN(3, "hipMallocPitch(");
    if (!rcuContext) createRcuContext();
    err = hipMallocPitch((void**)&devadr, &pitch, width, height);
    res.devPtr = (RCadr)devadr;
    res.pitch = pitch;
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, %d) done. devadr:0x%08llx\n", &devadr, width, height, devadr);

    return &res;
}

dscudaMemcpy2DToArrayH2HResult *
dscudamemcpy2dtoarrayh2hid_1_svc(RCadr dst, RCsize wOffset, RCsize hOffset, RCbuf src, RCsize spitch, RCsize width, RCsize height, struct svc_req *sr)
{
    static dscudaMemcpy2DToArrayH2HResult res;
    WARN(0, "dscudaMemcpy2DToArray() does not support hipMemcpyHostToHost transfer yet.\n");
    return &res;
}

dscudaResult *
dscudamemcpy2dtoarrayh2did_1_svc(RCadr dst, RCsize wOffset, RCsize hOffset, RCbuf srcbuf, RCsize spitch, RCsize width, RCsize height, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipMemcpy2DToArray(");
    if (!rcuContext) createRcuContext();
    err = hipMemcpy2DToArray((hipArray*)dst, wOffset, hOffset, srcbuf.RCbuf_val, spitch, width, height, hipMemcpyHostToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08lx, %d, %d, 0x%08llx, %d, %d, %d, %s) done.\n",
         dst, wOffset, hOffset, (unsigned long)srcbuf.RCbuf_val, spitch, width, height, dscudaMemcpyKindName(hipMemcpyHostToDevice));
    return &res;
}

dscudaMemcpy2DToArrayD2HResult *
dscudamemcpy2dtoarrayd2hid_1_svc(RCsize wOffset, RCsize hOffset, RCadr src, RCsize spitch, RCsize width, RCsize height, struct svc_req *sr)
{
    static RCsize maxcount = 0;
    hipError_t err;
    static dscudaMemcpy2DToArrayD2HResult res;
    int count = spitch * height;

    WARN(3, "hipMemcpy2DToArray(");
    if (!rcuContext) createRcuContext();
    if (maxcount == 0) {
        res.buf.RCbuf_val = NULL;
    }
    if (maxcount < count) {
        res.buf.RCbuf_val = (char*)realloc(res.buf.RCbuf_val, count);
        maxcount = count;
    }
    res.buf.RCbuf_len = count;
    err = hipMemcpy2DToArray((hipArray *)res.buf.RCbuf_val, wOffset, hOffset, (void *)src, spitch, width, height, hipMemcpyDeviceToHost);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, %d, 0x%08llx, %d, %d, %d, %s) done. 2D buf size : %d\n",
         (unsigned long)res.buf.RCbuf_val, wOffset, hOffset, src, spitch, width, height, dscudaMemcpyKindName(hipMemcpyDeviceToHost), count);
    return &res;
}

dscudaResult *
dscudamemcpy2dtoarrayd2did_1_svc(RCadr dst, RCsize wOffset, RCsize hOffset, RCadr src, RCsize spitch, RCsize width, RCsize height, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;
    WARN(3, "hipMemcpy2DToArray(");
    err = hipMemcpy2DToArray((hipArray *)dst, wOffset, hOffset, (void *)src, spitch, width, height, hipMemcpyDeviceToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, %d, 0x%08llx, %d, %d, %d, %s) done.\n",
         dst, wOffset, hOffset, src, spitch, width, height, dscudaMemcpyKindName(hipMemcpyDeviceToDevice));
    return &res;
}

dscudaMemcpy2DH2HResult *
dscudamemcpy2dh2hid_1_svc(RCadr dst, RCsize dpitch, RCbuf src, RCsize spitch, RCsize width, RCsize height, struct svc_req *sr)
{
    static dscudaMemcpy2DH2HResult res;
    WARN(0, "dscudaMemcpy2D() does not support hipMemcpyHostToHost transfer yet.\n");
    return &res;
}

dscudaResult *
dscudamemcpy2dh2did_1_svc(RCadr dst, RCsize dpitch, RCbuf srcbuf, RCsize spitch, RCsize width, RCsize height, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipMemcpy2D(");
    if (!rcuContext) createRcuContext();
    err = hipMemcpy2D((void*)dst, dpitch, srcbuf.RCbuf_val, spitch, width, height, hipMemcpyHostToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08lx, %d, 0x%08lx, %d, %d, %d, %s) done.\n",
         dst, dpitch, (unsigned long)srcbuf.RCbuf_val, spitch, width, height, dscudaMemcpyKindName(hipMemcpyHostToDevice));
    return &res;
}

dscudaMemcpy2DD2HResult *
dscudamemcpy2dd2hid_1_svc(RCsize dpitch, RCadr src, RCsize spitch, RCsize width, RCsize height, struct svc_req *sr)
{
    static RCsize maxcount = 0;
    hipError_t err;
    static dscudaMemcpy2DD2HResult res;
    int count = spitch * height;

    WARN(3, "hipMemcpy2D(");
    if (!rcuContext) createRcuContext();
    if (maxcount == 0) {
        res.buf.RCbuf_val = NULL;
    }
    if (maxcount < count) {
        res.buf.RCbuf_val = (char*)realloc(res.buf.RCbuf_val, count);
        maxcount = count;
    }
    res.buf.RCbuf_len = count;
    err = hipMemcpy2D(res.buf.RCbuf_val, dpitch, (void *)src, spitch, width, height, hipMemcpyDeviceToHost);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, 0x%08llx, %d, %d, %d, %s) done. 2D buf size : %d\n",
         (unsigned long)res.buf.RCbuf_val, dpitch, src, spitch, width, height, dscudaMemcpyKindName(hipMemcpyDeviceToHost), count);
    return &res;
}

dscudaResult *
dscudamemcpy2dd2did_1_svc(RCadr dst, RCsize dpitch, RCadr src, RCsize spitch, RCsize width, RCsize height, struct svc_req *sr)
{
    hipError_t err;
    static dscudaResult res;
    WARN(3, "hipMemcpy2D(");
    err = hipMemcpy2D((void *)dst, dpitch, (void *)src, spitch, width, height, hipMemcpyDeviceToDevice);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, 0x%08llx, %d, %d, %d, %s) done.\n",
         dst, dpitch, src, spitch, width, height, dscudaMemcpyKindName(hipMemcpyDeviceToDevice));
    return &res;
}

dscudaResult *
dscudamemset2did_1_svc(RCadr dst, RCsize pitch, int value, RCsize width, RCsize height, struct svc_req *sq)
{
    static dscudaResult res;
    hipError_t err;

    WARN(3, "hipMemset2D(");
    if (!rcuContext) createRcuContext();
    err = hipMemset2D((void *)dst, pitch, value, width, height);
    check_cuda_error(err);
    res.err = err;
    WARN(3, "0x%08llx, %d, %d, %d, %d) done.\n", dst, pitch, value, width, height);
    return &res;
}


/*
 * Texture Reference Management
 */

dscudaCreateChannelDescResult *
dscudacreatechanneldescid_1_svc(int x, int y, int z, int w, RCchannelformat f, struct svc_req *sr)
{
    static dscudaCreateChannelDescResult res;
    hipChannelFormatDesc desc;

    WARN(3, "hipCreateChannelDesc(");
    if (!rcuContext) createRcuContext();
    desc = hipCreateChannelDesc(x, y, z, w, (enum hipChannelFormatKind)f);
    res.x = desc.x;
    res.y = desc.y;
    res.z = desc.z;
    res.w = desc.w;
    res.f = desc.f;
    WARN(3, "%d, %d, %d, %d, %d) done.\n", x, y, z, w, f)
    return &res;
}

dscudaGetChannelDescResult *
dscudagetchanneldescid_1_svc(RCadr array, struct svc_req *sr)
{
    static dscudaGetChannelDescResult res;
    hipError_t err;
    hipChannelFormatDesc desc;

    WARN(3, "hipGetChannelDesc(");
    if (!rcuContext) createRcuContext();
    err = hipGetChannelDesc(&desc, (const struct hipArray*)array);
    res.err = err;
    res.x = desc.x;
    res.y = desc.y;
    res.z = desc.z;
    res.w = desc.w;
    res.f = desc.f;
    WARN(3, "0x%08llx, 0x&08llx) done.\n", &desc, array)
    return &res;
}

dscudaBindTextureResult *
dscudabindtextureid_1_svc(int moduleid, char *texname, RCadr devPtr, RCsize size, RCtexture texbuf, struct svc_req *sr)
{
    static dscudaBindTextureResult res;
    hipError_t err;
    hipTexRef texref;
    Module *mp = Modulelist + moduleid;

    if (!rcuContext) createRcuContext();

    err = (hipError_t)hipModuleGetTexRef(&texref, mp->handle, texname);
    WARN(3, "hipModuleGetTexRef(0x%08llx, 0x%08llx, %s) : module: %s\n",
         &texref, mp->handle, texname, mp->name);
    if (err != hipSuccess) {
        check_cuda_error(err);
        res.err = err;
        return &res;
    }

    err = setTextureParams(texref, texbuf, texname);
    if (err != hipSuccess) {
        res.err = err;
        return &res;
    }

    WARN(4, "hipTexRefSetAddress(0x%08llx, 0x%08llx, 0x%08llx, %d)\n", &res.offset, texref, devPtr, size);
    err = (hipError_t)hipTexRefSetAddress((size_t *)&res.offset, texref, (hipDeviceptr_t)devPtr, size);
    if (err != hipSuccess) {
        check_cuda_error(err);
        res.err = err;
        return &res;
    }
    res.err = err;

    return &res;
}

dscudaBindTexture2DResult *
dscudabindtexture2did_1_svc(int moduleid, char *texname, RCadr devPtr, RCsize width, RCsize height, RCsize pitch, RCtexture texbuf, struct svc_req *sr)
{
    static dscudaBindTexture2DResult res;
    hipError_t err;
    hipTexRef texref;
    Module *mp = Modulelist + moduleid;
    HIP_ARRAY_DESCRIPTOR desc;

    if (!rcuContext) createRcuContext();

    err = (hipError_t)hipModuleGetTexRef(&texref, mp->handle, texname);
    WARN(3, "hipModuleGetTexRef(0x%08llx, 0x%08llx, %s) : module: %s\n",
         &texref, mp->handle, texname, mp->name);
    if (err != hipSuccess) {
        check_cuda_error(err);
        res.err = err;
        return &res;
    }

    err = setTextureParams(texref, texbuf, texname, &desc);
    if (err != hipSuccess) {
        res.err = err;
        return &res;
    }
    desc.Height = height;
    desc.Width  = width;

    WARN(4, "hipTexRefSetAddress2D(0x%08llx, 0x%08llx, 0x%08llx, %d)\n", texref, desc, devPtr, pitch);
    err = (hipError_t)hipTexRefSetAddress2D(texref, &desc, (hipDeviceptr_t)devPtr, pitch);
    if (err != hipSuccess) {
        check_cuda_error(err);
        res.err = err;
        return &res;
    }
    res.err = err;

    unsigned int align = hipDeviceAttributeTextureAlignment;
    unsigned long int roundup_adr = ((devPtr - 1) / align + 1) * align;
    res.offset = roundup_adr - devPtr;
    WARN(4, "align:0x%x  roundup_adr:0x%08llx  devPtr:0x%08llx  offset:0x%08llx\n",
         align, roundup_adr, devPtr, res.offset);
    return &res;
}

dscudaResult *
dscudabindtexturetoarrayid_1_svc(int moduleid, char *texname, RCadr array, RCtexture texbuf, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err;
    hipTexRef texref;
    Module *mp = Modulelist + moduleid;

    if (!rcuContext) createRcuContext();

    err = (hipError_t)hipModuleGetTexRef(&texref, mp->handle, texname);
    WARN(3, "hipModuleGetTexRef(0x%08llx, 0x%08llx, %s) : module: %s  moduleid:%d\n",
         &texref, mp->handle, texname, mp->name, moduleid);
    if (err != hipSuccess) {
        check_cuda_error(err);
        res.err = err;
        return &res;
    }

    err = setTextureParams(texref, texbuf, texname);
    if (err != hipSuccess) {
        res.err = err;
        return &res;
    }

    WARN(4, "hipTexRefSetArray(0x%08llx, 0x%08llx, %d)\n", texref, array, HIP_TRSA_OVERRIDE_FORMAT);
    err = (hipError_t)hipTexRefSetArray(texref, (hipArray_t)array, HIP_TRSA_OVERRIDE_FORMAT);
    if (err != hipSuccess) {
        check_cuda_error(err);
        res.err = err;
        return &res;
    }
    res.err = err;
    return &res;
}

dscudaResult *
dscudaunbindtextureid_1_svc(RCtexture texrefbuf, struct svc_req *sr)
{
    static dscudaResult res;
    hipError_t err = hipSuccess;

    WARN(4, "Current implementation of hipUnbindTexture() does nothing "
         "but returning hipSuccess.\n");

    res.err = err;
    return &res;
}

dscudaLoadModuleResult *
dscudaloadmoduleid_1_svc(RCipaddr ipaddr, RCpid pid, char *mname, char *image, struct svc_req *sr)
{
    static dscudaLoadModuleResult res;
    res.id = dscudaLoadModule(ipaddr, pid, mname, image);
    return &res;
}

/*
 * launch a kernel function of id 'kid' (or name 'kname', if it's not loaded yet),
 * defined in a module of id 'moduleid'.
 */
void *
dscudalaunchkernelid_1_svc(int moduleid, int kid, char *kname,
                          RCdim3 gdim, RCdim3 bdim, RCsize smemsize, RCstream stream, RCargs args, struct svc_req *sr)
{
    static int dummyres = 0;
    dscudaLaunchKernel(moduleid, kid, kname, gdim, bdim, smemsize, stream, args);
    return &dummyres; // seems necessary to return something even if it's not used by the client.
}

