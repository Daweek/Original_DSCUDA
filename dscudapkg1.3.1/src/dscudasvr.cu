#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include <unistd.h>
#include <rpc/rpc.h>
#include <rpc/pmap_clnt.h>
#include <cutil.h>
// remove definition of some macros which will be redefined in \"cutil_inline.h\".
#ifdef MIN
#undef MIN
#endif
#ifdef MAX
#undef MAX
#endif
#include <cutil_inline.h>
#include <hipfft/hipfft.h>
#include <pthread.h>
#include <sys/types.h>
#include <sys/stat.h>
#include <poll.h>
#include <errno.h>

#include "dscudarpc.h"
#include "dscuda.h"
#include "sockutil.h"
#include "ibv_rdma.h"

typedef struct {
    int valid;
    unsigned int id;
    unsigned int ipaddr;
    unsigned int pid;
    time_t loaded_time;
    char name[256];
    hipModule_t handle;
    hipFunction_t kfunc[RC_NKFUNCMAX]; // this is not used for now.
} Module;

static int D2Csock = -1; // socket for sideband communication to the client. inherited from the daemon.
static int TcpPort = RC_SERVER_IP_PORT;
static int Connected = 0;
static int UseIbv = 0; // use IB Verbs if set to 1. use RPC by default.
static int Ndevice = 1;                 // # of devices in the system.
static int Devid[RC_NDEVICEMAX] = {0,}; // real device ids of the ones in the system.
static int dscuDevice;                   // virtual device id of the one used in the current context.
static hipCtx_t dscuContext = NULL;
static int Devid2Vdevid[RC_NDEVICEMAX]; // device id conversion table from real to virtual.
static Module Modulelist[RC_NKMODULEMAX] = {0};

static void notifyIamReady(void);
static void showUsage(char *command);
static void showConf(void);
static void parseArgv(int argc, char **argv);
static hipError_t initDscuda(void);
static hipError_t createDscuContext(void);
static hipError_t destroyDscuContext(void);
static void initEnv(void);
static void releaseModules(bool releaseall);
static hipError_t getFunctionByName(hipFunction_t *kfuncp, char *kname, int moduleid);
static void getGlobalSymbol(int moduleid, char *symbolname, hipDeviceptr_t *dptr, size_t *size);
static int dscudaLoadModule(RCipaddr ipaddr, RCpid pid, char *mname, char *image);
static void *dscudaLaunchKernel(int moduleid, int kid, char *kname, RCdim3 gdim, RCdim3 bdim, RCsize smemsize, RCstream stream, RCargs args);
static hipError_t setTextureParams(hipTexRef texref, RCtexture texbuf, char *texname, HIP_ARRAY_DESCRIPTOR *descp = NULL);

#undef WARN
#define WARN(lv, fmt, args...) if (lv <= dscudaWarnLevel()) \
        fprintf(stderr, "dscudasvr[%d] : " fmt, TcpPort - RC_SERVER_IP_PORT, ## args);

#if 0
inline void
fatal_error(int exitcode)
{
    fprintf(stderr,
            "%s(%i) : fatal_error().\n"
            "Probably you need to restart dscudasvr.\n",
            __FILE__, __LINE__);
    exit(exitcode);
}

inline void
check_cuda_error(hipError_t err)
{
    if (hipSuccess != err) {
        fprintf(stderr,
                "%s(%i) : check_cuda_error() Runtime API error : %s.\n"
                "You may need to restart dscudasvr.\n",
                __FILE__, __LINE__, hipGetErrorString(err));
    }
}
#else

#define fatal_error(exitcode)\
{\
    fprintf(stderr,\
            "%s(%i) : fatal_error().\n"\
            "Probably you need to restart dscudasvr.\n",\
            __FILE__, __LINE__);\
    exit(exitcode);\
}\

#define check_cuda_error(err)\
{\
    if (hipSuccess != err) {\
        fprintf(stderr,\
                "%s(%i) : check_cuda_error() Runtime API error : %s.\n"\
                "You may need to restart dscudasvr.\n",\
                __FILE__, __LINE__, hipGetErrorString(err));\
    }\
}
#endif

#if !RPC_ONLY
#include "dscudasvr_ibv.cu"
#endif
#include "dscudasvr_rpc.cu"


static void
notifyIamReady(void)
{
    char msg[] = "ready";
    if (D2Csock >= 0) {
        WARN(3, "send \"ready\" to the client.\n");
        sendMsgBySocket(D2Csock, msg);
    }
}

static int
receiveProtocolPreference(void)
{
    char msg[256], rc[64];

    if (D2Csock >= 0) {
        WARN(3, "wait for remotecall preference (\"rpc\" or \"ibv\") from the client.\n");
        recvMsgBySocket(D2Csock, msg, sizeof(msg));
        sscanf(msg, "remotecall:%s", rc);
        WARN(2, "method of remote procedure call: %s\n", rc);
        if (!strncmp("ibv", rc, strlen("ibv"))) {
            return 1;
        }
        else {
            return 0;
        }
    }
    else {
        return UseIbv; // do not modify the preference.
    }
}

int
main(int argc, char **argv)
{
    parseArgv(argc, argv);
    initEnv();
    initDscuda();
    showConf();

    UseIbv = receiveProtocolPreference();

    if (UseIbv) {
#if !RPC_ONLY
        setupIbv();
        notifyIamReady();
        ibvMainLoop(NULL);
#endif
    }
    else {
        setupRpc();
        notifyIamReady();
        svc_run(); // RPC main loop.
    }
    fprintf (stderr, "main loop returned.\n"); // never reached.
    exit (1);
}

static void
showUsage(char *command)
{
    fprintf(stderr,
            "usage: %s [-s server_id] [-d 'deviceid'] [-p port] [-S socket]\n"
            "       (-p & -S are used by the daemon only.)\n",
            command);
}

static void
showConf(void)
{
    int i;
    char str[1024], str0[1024];

    WARN(2, "TCP port : %d (base + %d)\n", TcpPort, TcpPort - RC_SERVER_IP_PORT);
    WARN(2, "ndevice : %d\n", Ndevice);
    sprintf(str, "real device%s      :", Ndevice > 1 ? "s" : " ");
    for (i = 0; i < Ndevice; i++) {
        sprintf(str0, " %d", Devid[i]);
        strcat(str, str0);
    }
    WARN(2, "%s\n", str);
    sprintf(str, "virtual device%s   :", Ndevice > 1 ? "s" : " ");
    for (i = 0; i < Ndevice; i++) {
        sprintf(str0, " %d", Devid2Vdevid[Devid[i]]);
        strcat(str, str0);
    }
    WARN(2, "%s\n", str);
}

extern char *optarg;
extern int optind;
static void
parseArgv(int argc, char **argv)
{
    int c, ic;
    char *param = "d:hp:s:S:";
    char *num;
    char buf[256];
    int device_used[RC_NDEVICEMAX] = {0,};
    int tcpport_set = 0;
    int serverid = 0;

    while ((c = getopt(argc, argv, param)) != EOF) {
        switch (c) {
          case 'p':
            TcpPort = atoi(optarg);
            tcpport_set = 1;
            break;

          case 's':
            serverid = atoi(optarg);
            break;

          case 'S':
            D2Csock = atoi(optarg);
            break;

          case 'd':
            Ndevice = 0;
            strncpy(buf, optarg, sizeof(buf));
            num = strtok(buf, " ");
            while (num) {
                ic = atoi(num);
                if (ic < 0 || RC_NDEVICEMAX <= ic ) {
                    fprintf(stderr, "device id out of range: %d\n", ic);
                    exit(2);
                }
                if (!device_used[ic]) { // care for malformed optarg value such as "0 1 2 2 3".
                    device_used[ic] = 1;
                    Devid[Ndevice] = ic;
                    Ndevice++;
                }
                num = strtok(NULL, " ");
            }
            break;

          case 'h':
          default:
            showUsage(argv[0]);
            exit(1);
        }
    }
    if (!tcpport_set) {
        TcpPort = RC_SERVER_IP_PORT + serverid;
        WARN(3, "TCP port number not given by '-p' option. Use default (%d).\n", TcpPort);
    }
}

// should be called only once in a run.
static hipError_t
initDscuda(void)
{
    int i;
    unsigned int flags = 0; // should always be 0.
    hipError_t err;

    WARN(4, "initDscuda...\n");

    for (i = 0; i < Ndevice; i++) {
        Devid2Vdevid[Devid[i]] = i;
    }

    err = hipInit(flags);
    if (err != hipSuccess) {
        WARN(0, "hipInit(%d) failed.\n", flags);
        exit(1);
    }
    err = (hipError_t)hipSetValidDevices(Devid, Ndevice);
    if (err != hipSuccess) {
        WARN(0, "hipSetValidDevices(0x%08llx, %d) failed.\n", Devid, Ndevice);
        exit(1);
    }
    dscuDevice = Devid[0];
    WARN(3, "hipSetValidDevices(0x%08llx, %d). dscuDevice:%d\n",
         Devid, Ndevice, dscuDevice);
    WARN(4, "initDscuda done.\n");
    return (hipError_t)err;
}

static hipError_t
createDscuContext(void)
{
    //    unsigned int flags = 0; // should always be 0.
    hipDevice_t dev = 0;
    hipError_t err;

    err = hipDeviceGet(&dev, dscuDevice);
    if (err != hipSuccess) {
        WARN(0, "hipDeviceGet() failed.\n");
        return (hipError_t)err;
    }

#if 0
    err = hipCtxCreate(&dscuContext, flags, dev);
    if (err != hipSuccess) {
        WARN(0, "hipCtxCreate() failed.\n");
        return (hipError_t)err;
    }
#else // not used. set a dummy value not to be called repeatedly.
    dscuContext = (hipCtx_t)-1;
#endif

    return (hipError_t)err;
}

static hipError_t
destroyDscuContext(void)
{
#if 0

    hipError_t cuerr;
    bool all = true;

    WARN(3, "destroyDscuContext(");
    releaseModules(all);

    cuerr = hipCtxDestroy(dscuContext);
    WARN(4, "hipCtxDestroy(0x%08llx", dscuContext);
    if (cuerr != hipSuccess) {
        WARN(0, "hipCtxDestroy() failed.\n");
        fatal_error(1);
        return (hipError_t)cuerr;
    }
    dscuContext = NULL;
    WARN(4, ") done.\n");
    WARN(3, ") done.\n");

#else

    dscuContext = NULL;

#endif
    return hipSuccess;
}

static void
initEnv(void)
{
    static int firstcall = 1;
    char *env;

    if (!firstcall) return;

    firstcall = 0;

    // DSCUDA_WARNLEVEL
    env = getenv("DSCUDA_WARNLEVEL");
    if (env) {
        int tmp;
        tmp = atoi(strtok(env, " "));
        if (0 <= tmp) {
            dscudaSetWarnLevel(tmp);
        }
        WARN(1, "WarnLevel: %d\n", dscudaWarnLevel());
    }

    // DSCUDA_REMOTECALL
    env = getenv("DSCUDA_REMOTECALL");
#if RPC_ONLY
    UseIbv = 0;
    WARN(2, "method of remote procedure call: RPC\n");
#else
    if (!env) {
        fprintf(stderr, "Set an environment variable 'DSCUDA_REMOTECALL' to 'ibv' or 'rpc'.\n");
        exit(1);
    }
    if (!strcmp(env, "ibv")) {
        UseIbv = 1;
        WARN(2, "method of remote procedure call: InfiniBand Verbs\n");
    }
    else if (!strcmp(env, "rpc")) {
        UseIbv = 0;
        WARN(2, "method of remote procedure call: RPC\n");
    }
    else {
        UseIbv = 0;
        WARN(2, "method of remote procedure call '%s' is not available. use RPC.\n", env);
    }
#endif
}

/*
 * Unload Modules never been used for a long time.
 */
static void
releaseModules(bool releaseall = false)
{
    Module *mp;
    int i;

    for (i = 0, mp = Modulelist; i < RC_NKMODULEMAX; i++, mp++) {
        if (!mp->valid) continue;
        if (releaseall || time(NULL) - mp->loaded_time > RC_SERVER_CACHE_LIFETIME) {
            hipModuleUnload((hipModule_t)mp->handle);
            mp->valid = 0;
            mp->handle = NULL;
            for (i = 0; i < RC_NKFUNCMAX; i++) {
                mp->kfunc[i] = NULL;
            }
            WARN(3, "releaseModules() unloaded a module. name:%s pid:%d ip:%s age:%d\n",
                 mp->name, mp->pid, dscudaGetIpaddrString(mp->ipaddr),
                 time(NULL) - mp->loaded_time);
        }
    }
}

static hipError_t
getFunctionByName(hipFunction_t *kfuncp, char *kname, int moduleid)
{
    hipError_t cuerr;
    Module *mp = Modulelist + moduleid;

    cuerr = hipModuleGetFunction(kfuncp, mp->handle, kname);
    if (cuerr == hipSuccess) {
        WARN(3, "hipModuleGetFunction() : function '%s' found.\n", kname);
    }
    else {
        WARN(0, "hipModuleGetFunction() : function:'%s'. %s\n",
             kname, hipGetErrorString((hipError_t)cuerr));
	WARN(0, "moduleid:%d module valid:%d id:%d name:%s\n",
	     moduleid, mp->valid, mp->id, mp->name);
        fatal_error(1);
    }
    return cuerr;
}

static void
getGlobalSymbol(int moduleid, char *symbolname, hipDeviceptr_t *dptr, size_t *size)
{
    hipError_t cuerr;
    Module *mp;

    if (moduleid < 0 || RC_NKMODULEMAX <= moduleid) {
        WARN(0, "getGlobalSymbol() : invalid module id:%d.\n", moduleid);
        fatal_error(1);
    }
    mp = Modulelist + moduleid;
    cuerr = hipModuleGetGlobal(dptr, size, mp->handle, symbolname);
    if (cuerr == hipSuccess) {
    WARN(3, "hipModuleGetGlobal(0x%08lx, 0x%08lx, 0x%08lx, %s) done."
	 " modulename:%s  symbolname:%s  *dptr:0x%08lx\n",
	 dptr, size, mp->handle, symbolname,
	 mp->name, symbolname, *dptr);
    }
    else {
        WARN(0, "hipModuleGetGlobal(0x%08llx, 0x%08llx, 0x%08llx, 0x%08llx) failed."
             " modulename:%s  symbolname:%s  %s\n",
             dptr, size, mp->handle, symbolname,
             mp->name, symbolname, hipGetErrorString((hipError_t)cuerr));
        fatal_error(1);
    }
}

static int
dscudaLoadModule(RCipaddr ipaddr, RCpid pid, char *mname, char *image)
{
    hipError_t cuerr;
    Module   *mp;
    int      i;

#if RC_CACHE_MODULE
    // look for mname in the module list, which may found if the client
    // resent multiple requests for the same mname:pid:ipaddr.
    int found = 0;
    for (i = 0, mp = Modulelist; i < RC_NKMODULEMAX; i++, mp++) {
        if (!mp->valid) continue;
        if ((unsigned int)ipaddr == mp->ipaddr &&
            pid    == mp->pid &&
            !strcmp(mname, mp->name)) {
            found = 1;
            break;
        }
	WARN(4, "ip:%x  %x    pid:%d  %d    name:%s  %s\n",
	     (unsigned int)ipaddr, mp->ipaddr, pid, mp->pid, mname, mp->name);
    }

    if (found) { // module found. i.e, it's already loaded.
        WARN(3, "\n\n------------------------------------------------------------------\n"
             "dscudaloadmoduleid_1_svc() got multiple requests for\n"
             "  the same module name : %s,\n"
             "  the same process id  : %d, and\n"
             "  the same IP address  : %s,\n"
             "which means a client resent the same module twice or more.\n"
             "If you see this message too often, you may want to increase\n"
             "$dscuda/include/dscudadefs.h:RC_CLIENT_CACHE_LIFETIME\n"
             "for better performance.\n"
             "------------------------------------------------------------------\n\n",
             mname, pid, dscudaGetIpaddrString(ipaddr));
        WARN(3, "hipModuleLoadData() : a module found in the cache. id:%d  name:%s  age:%d\n",
             mp->id, mname, time(NULL) - mp->loaded_time);
    }
    else  // module not found in the cache. load it from image.
#endif // RC_CACHE_MODULE

    {
        for (i = 0, mp = Modulelist; i < RC_NKMODULEMAX; i++, mp++) {
            if (!mp->valid) break;
            if (i == RC_NKMODULEMAX) {
                WARN(0, "module cache is full.\n");
                fatal_error(1);
            }
        }
        mp->id = i;
        cuerr = hipModuleLoadData(&mp->handle, image);
        if (cuerr != hipSuccess) {
            WARN(0, "hipModuleLoadData() failed. %s\n", hipGetErrorString((hipError_t)cuerr));
            fatal_error(1);
        }
        mp->valid = 1;
        mp->ipaddr = ipaddr;
        mp->pid = pid;
        strncpy(mp->name, mname, sizeof(Modulelist[0].name));
        for (i = 0; i < RC_NKFUNCMAX; i++) {
            mp->kfunc[i] = NULL;
        }
        WARN(3, "hipModuleLoadData() : a module loaded. id:%d  name:%s\n", mp->id, mname);
    }
    mp->loaded_time = time(NULL); // (re)set the lifetime of the cache.
    releaseModules();
    return mp->id;
}

static void *
dscudaLaunchKernel(int moduleid, int kid, char *kname,
                   RCdim3 gdim, RCdim3 bdim, RCsize smemsize, RCstream stream, RCargs args)
{
    static int dummyres = 123;
    int paramsize;
    hipError_t cuerr;

#if !RC_SUPPORT_CONCURRENT_EXEC
    stream = 0;
#endif

    if (!dscuContext) createDscuContext();

    // load a kernel function into Module[moduleid].kfunc[kid]
    // form Module[moduleid].handle.
    if (moduleid < 0 || RC_NKMODULEMAX <= moduleid) {
        WARN(0, "dscudalaunchkernelid_1_svc() : invalid module id:%d.\n", moduleid);
        fatal_error(1);
    }

#if 1 // look inside a module for a function by name.
    hipFunction_t kfunc;
    getFunctionByName(&kfunc, kname, moduleid);
#else // look for a function by its ID.
    // this is faster, but not used since it would cause a problem
    // when called from a kernel function that uses C++ template.
    // in that case kid might not be unique for each instance of the template.
    Module *mp = Modulelist + moduleid;
    hipFunction_t kfunc = mp->kfunc[kid];
    if (!kfunc) {
        getFunctionByName(&kfunc, kname, moduleid);
        mp->kfunc[kid] = kfunc;
    }
#endif

    // a kernel function found.
    // now make it run.
    if (UseIbv) {
#if !RPC_ONLY
        paramsize = ibvUnpackKernelParam(&kfunc, args.RCargs_len, (IbvArg *)args.RCargs_val);
#endif
    }
    else {
        paramsize = rpcUnpackKernelParam(&kfunc, &args);
    }
    cuerr = cuParamSetSize(kfunc, paramsize);
    if (cuerr != hipSuccess) {
        WARN(0, "cuParamSetSize() failed. size:%d %s\n",
             paramsize, hipGetErrorString((hipError_t)cuerr));
        fatal_error(1);
    }
    WARN(5, "cuParamSetSize() done.\n");

    cuerr = cuFuncSetBlockShape(kfunc, bdim.x, bdim.y, bdim.z);
    if (cuerr != hipSuccess) {
        WARN(0, "cuFuncSetBlockShape() failed. %s\n", hipGetErrorString((hipError_t)cuerr));
        fatal_error(1);
    }
    WARN(5, "cuFuncSetBlockShape() done.\n");

    if (smemsize != 0) {
        cuerr = cuFuncSetSharedSize(kfunc, smemsize);
        if (cuerr != hipSuccess) {
            WARN(0, "cuFuncSetSharedSize() failed. %s\n", hipGetErrorString((hipError_t)cuerr));
            fatal_error(1);
        }
        WARN(5, "cuFuncSetSharedSize() done.\n");
    }

    if ((hipStream_t)stream == NULL) {
        cuerr = cuLaunchGrid(kfunc, gdim.x, gdim.y);
        if (cuerr != hipSuccess) {
            WARN(0, "cuLaunchGrid() failed. kname:%s %s\n",
                 kname, hipGetErrorString((hipError_t)cuerr));
            fatal_error(1);
        }
        WARN(3, "cuLaunchGrid() done. kname:%s\n", kname);
    }
    else {
        cuerr = cuLaunchGridAsync(kfunc, gdim.x, gdim.y, (hipStream_t)stream);
        if (cuerr != hipSuccess) {
            WARN(0, "cuLaunchGridAsync() failed. kname:%s  %s\n",
                 kname, hipGetErrorString((hipError_t)cuerr));
            fatal_error(1);
        }
        WARN(3, "cuLaunchGridAsync() done.  kname:%s  stream:0x%08llx\n", kname, stream);
    }

    return &dummyres; // seems necessary to return something even if it's not used by the client.
}

static hipError_t
setTextureParams(hipTexRef texref, RCtexture texbuf, char *texname, HIP_ARRAY_DESCRIPTOR *descp)
{
    hipError_t err;
    int ncomponent, i;
    unsigned int texref_flags = 0;
    unsigned int fmt_high, fmt_low, fmt_index;

    hipArray_Format fmt[] = {
        HIP_AD_FORMAT_UNSIGNED_INT8,    // 00_00
        HIP_AD_FORMAT_UNSIGNED_INT16,   // 00_01
        HIP_AD_FORMAT_UNSIGNED_INT32,   // 00_10
        (hipArray_Format)0,
        HIP_AD_FORMAT_SIGNED_INT8,      // 01_00
        HIP_AD_FORMAT_SIGNED_INT16,     // 01_01
        HIP_AD_FORMAT_SIGNED_INT32,     // 01_10
        (hipArray_Format)0,
        (hipArray_Format)0,
        HIP_AD_FORMAT_HALF,             // 10_01
        (hipArray_Format)0,
        (hipArray_Format)0,
        (hipArray_Format)0,
        (hipArray_Format)0,
        HIP_AD_FORMAT_FLOAT,            // 11_10
        (hipArray_Format)0,
    };

    // set addressmode (wrap/clamp/mirror/border)
    //
    for (i = 0; i < 3; i++) {
        err = (hipError_t)hipTexRefSetAddressMode(texref, i, (HIPaddress_mode_enum)texbuf.addressMode[i]);
        if (err != hipSuccess) {
            check_cuda_error(err);
            return err;
        }
    }

    // set filtermode (point/linear)
    //
    err = (hipError_t)hipTexRefSetFilterMode(texref, (HIPfilter_mode_enum)texbuf.filterMode);
    if (err != hipSuccess) {
        check_cuda_error(err);
        return err;
    }

    // set flags (integer/normalized)
    //
    if (texbuf.normalized) {
        texref_flags |= HIP_TRSF_NORMALIZED_COORDINATES;
    }
    else {
        texref_flags |= HIP_TRSF_READ_AS_INTEGER;
    }
    err = (hipError_t)hipTexRefSetFlags(texref, texref_flags);
    if (err != hipSuccess) {
        check_cuda_error(err);
        return err;
    }

    // set format (unsigned/signed/float, 32/16/8-bit)
    //
    switch (texbuf.x) {
      case 8:
        fmt_low = 0;
        break;
      case 16:
        fmt_low = 1;
        break;
      case 32:
        fmt_low = 2;
        break;
      default:
        WARN(0, "hipModuleGetTexRef() invalid channel format. texture name:%s descriptor.x:%d\n",
             texname, texbuf.x);
        err = hipErrorInvalidValue;
        return err;
    }
    switch (texbuf.f) {
      case hipChannelFormatKindUnsigned:
        fmt_high = 0;
        break;

      case hipChannelFormatKindSigned:
        fmt_high = 1;
        break;

      case hipChannelFormatKindFloat:
        fmt_high = 3;
        break;

      case hipChannelFormatKindNone:
        WARN(0, "hipModuleGetTexRef() invalid channel format. texture name:%s descriptor.f:%s\n",
             texname, "hipChannelFormatKindNone");
        err = hipErrorInvalidValue;
        return err;

      default:
        WARN(0, "hipModuleGetTexRef() invalid channel format. texture name:%s descriptor.f:%s\n",
             texname, texbuf.f);
        err = hipErrorInvalidValue;
        return err;
    }
    fmt_index = fmt_high << 2 | fmt_low;
    ncomponent = 1;
    if (texbuf.y) ncomponent = 2;
    if (texbuf.z) ncomponent = 3;
    if (texbuf.w) ncomponent = 4;
    if (descp) {
        descp->Format = fmt[fmt_index];
        descp->NumChannels = ncomponent;
    }
    WARN(4, "hipTexRefSetFormat(0x%08llx, %d, %d)\n", texref, fmt[fmt_index], ncomponent);
    err = (hipError_t)hipTexRefSetFormat(texref, fmt[fmt_index], ncomponent);
    if (err != hipSuccess) {
        check_cuda_error(err);
        return err;
    }

    return hipSuccess;
}
