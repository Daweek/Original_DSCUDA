
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define N (8)

#define safeCall(err)             __safeCall   (err, __FILE__, __LINE__)

static inline void
__safeCall(hipError_t err, const char *file, const int line)
{
    if (hipSuccess != err) {
        fprintf(stderr, "%s(%i) : __unsafeCall() Runtime API error %d: %s.\n",
                file, line, (int)err, hipGetErrorString(err));
        exit(1);
    }
}

namespace foo {

__constant__ float MyVar0;

__global__ void
vecAdd(float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i] + MyVar0;
}

__global__ void
vecMul(float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

template <typename T0, typename T1> __global__ void
vecAddT(T1 *a, T1 *b, T0 *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

static int
local_func(void)
{
    int i, t;
    float floatvar;
    float a[N], b[N], c[N];
    float *d_a, *d_b, *d_c;
    double *dd_a, *dd_b, *dd_c;
    void (*func)(float *, float *, float *);
    float coeff;

    safeCall(hipMalloc((void**) &d_a, sizeof(float) * N));
    safeCall(hipMalloc((void**) &d_b, sizeof(float) * N));
    safeCall(hipMalloc((void**) &d_c, sizeof(float) * N));

    for (t = 0; t < 3; t++) {
        printf("try %d\n", t);
        for (i = 0; i < N; i++) {
            a[i] = rand()%64;
            b[i] = rand()%64;
            coeff = rand()%64;
        }
        safeCall(hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice));
        safeCall(hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice));

        //        fprintf(stderr, "MyVar0: 0x%016llx  &::MyVar0: 0x%016llx\n", MyVar0, &::MyVar0);

        safeCall(hipMemcpyToSymbol(HIP_SYMBOL(MyVar0), &coeff, sizeof(float), 0, hipMemcpyHostToDevice));
        safeCall(hipMemcpyFromSymbol(&floatvar, HIP_SYMBOL(MyVar0), sizeof(float), 0, hipMemcpyDeviceToHost));
        printf("floatvar:%f\n", floatvar);

        int nth = 4;
        dim3 threads(nth, 1, 1);
        dim3 grids((N+nth-1)/nth, 1, 1);
        vecAdd<<<grids, threads>>>(d_a, d_b, d_c);
        //        vecAddT<float, float><<<grids, threads>>>(d_a, d_b, d_c);
        //        vecAddT<float, double><<<grids, threads>>>(dd_a, dd_b, d_c);

        func = vecAdd;
        //        fprintf(stderr, ">>>>%x\n", func);
        safeCall(hipMemcpy(c, d_c, sizeof(float) * N, hipMemcpyDeviceToHost));
        for (i = 0; i < N; i++) {
            printf("% 6.2f + % 6.2f + % 6.2f = % 7.2f",
                   a[i], b[i], coeff, c[i]);
            if (a[i] + b[i] + coeff != c[i]) printf("   NG");
            printf("\n");
        }
        printf("\n");
    }

    exit(0);
}
}

int main(void)
{
    foo::local_func();
    exit(0);
}
