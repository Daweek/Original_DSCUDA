
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

__global__ void
vecAdd(float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

__global__ void
vecMul(float *a, float *b, float *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    c[i] = a[i] + b[i];
}

