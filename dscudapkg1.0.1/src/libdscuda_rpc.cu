#include "libdscuda.h"

static CLIENT *Clnt[RC_NVDEVMAX][RC_NREDUNDANCYMAX];

int
dscudaRemoteCallType(void)
{
    return RC_REMOTECALL_TYPE_RPC;
}

static void
setupConnection(int idev, RCServer_t *sp)
{
    int id = sp->id;
    int cid = sp->cid;
    int portid = DSCUDA_PROG + cid;

    WARN(2, "Requesting socket connection to %s:%d (port 0x%x)...\n", sp->ip, cid, portid);

#if 0
    Clnt[idev][id] = clnt_create(sp->ip, DSCUDA_PROG, DSCUDA_VER, "tcp");

#elif 1 // TCP

    struct sockaddr_in sockaddr;
    struct hostent *hent;
    int sock = RPC_ANYSOCK;

    hent = gethostbyname(sp->ip);
    memset((char *)&sockaddr, 0, sizeof(sockaddr));
    sockaddr.sin_family = AF_INET;
    bcopy(hent->h_addr, (caddr_t)&sockaddr.sin_addr, hent->h_length);
    sockaddr.sin_port = htons((in_port_t)0);

    Clnt[idev][id] = clnttcp_create(&sockaddr,
                                    portid,
                                    DSCUDA_VER,
                                    &sock,
                                    RC_BUFSIZE, RC_BUFSIZE);

#else // UDP

    struct sockaddr_in sockaddr;
    struct hostent *hent;
    int sock = RPC_ANYSOCK;

    hent = gethostbyname(sp->ip);
    memset((char *)&sockaddr, 0, sizeof(sockaddr));
    sockaddr.sin_family = AF_INET;
    bcopy(hent->h_addr, (caddr_t)&sockaddr.sin_addr, hent->h_length);
    sockaddr.sin_port = htons((in_port_t)0);
    struct timeval wait = {
        1.0, // sec
        0.0, // usec
    };

    Clnt[idev][id] = clntudp_create(&sockaddr,
                                    portid,
                                    DSCUDA_VER,
                                    wait,
                                    &sock);

#endif
    if (!Clnt[idev][id]) {
        char buf[256];
        sprintf(buf, "%s:%d (port 0x%x) ", sp->ip, id, portid);
        clnt_pcreateerror(buf);
        if (0 == strcmp(sp->ip, DEFAULT_SVRIP)) {
            WARN(0, "You may need to set an environment variable 'DSCUDA_SERVER'.\n");
        }
        else {
            WARN(0, "DSCUDA server (dscudasrv on %s:%d) may be down.\n", sp->ip, id);
        }
        exit(1);
    }
}

static void
checkResult(void *rp, RCServer_t *sp)
{
    if (rp) return;
    clnt_perror(Clnt[Vdevid][sp->id], sp->ip);
    exit(1);
}

/*
 * test API for internal use only.
 */

void
dscudaWrite(size_t size, char *dst, char *src)
{
    dscudaResult *rp;
    RCbuf srcbuf;

    initClient();
    srcbuf.RCbuf_len = size;
    srcbuf.RCbuf_val = (char *)src;
    WARN(3, "cudaWrite(%d, 0x%08llx, 0x%08llx)...", size, (unsigned long)dst, (unsigned long)src);

    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudawriteid_1((RCsize)size, (RCadr)dst, srcbuf, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
    }
    WARN(3, "done.\n");
}

void
dscudaRead(size_t size, char *dst, char *src)
{
    dscudaReadResult *rp;

    initClient();
    WARN(3, "cudaRead(%d, 0x%08llx, 0x%08llx)...", size, (unsigned long)dst, (unsigned long)src);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudareadid_1((RCsize)size, (RCadr)src, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
    }
    memcpy(dst, rp->buf.RCbuf_val, rp->buf.RCbuf_len);
    WARN(3, "done.\n");
}

/*
 * Dscuda client-side counterpart for CUDA runtime API:
 */

/*
 * Thread Management
 */

hipError_t
hipDeviceReset(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipDeviceReset()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadexitid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceSynchronize(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipDeviceSynchronize()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadsynchronizeid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
cudaThreadSetLimit(enum hipLimit_t limit, size_t value)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "cudaThreadSetLimit(%d, %d)...", limit, value);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadsetlimitid_1(limit, value, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
cudaThreadGetLimit(size_t *pValue, enum hipLimit_t limit)
{
    hipError_t err = hipSuccess;
    dscudaThreadGetLimitResult *rp;

    initClient();
    WARN(3, "cudaThreadGetLimit(0x%08llx, %d)...", pValue, limit);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadgetlimitid_1(limit, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    *pValue = rp->value;
    err = (hipError_t)rp->err;
    WARN(3, "done.  *pValue: %d\n", *pValue);

    return err;
}

hipError_t
hipDeviceSetCacheConfig(enum hipFuncCache_t cacheConfig)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipDeviceSetCacheConfig(%d)...", cacheConfig);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadsetcacheconfigid_1(cacheConfig, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceGetCacheConfig(enum hipFuncCache_t *pCacheConfig)
{
    hipError_t err = hipSuccess;
    dscudaThreadGetCacheConfigResult *rp;

    initClient();
    WARN(3, "hipDeviceGetCacheConfig(0x%08llx)...", pCacheConfig);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudathreadgetcacheconfigid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    *pCacheConfig = (enum hipFuncCache_t)rp->cacheConfig;
    err = (hipError_t)rp->err;
    WARN(3, "done.  *pCacheConfig: %d\n", *pCacheConfig);

    return err;
}


/*
 * Error Handling
 */

hipError_t
hipGetLastError(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(5, "hipGetLastError()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudagetlasterrorid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(5, "done.\n");

    return err;
}

hipError_t
hipPeekAtLastError(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(5, "hipPeekAtLastError()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudapeekatlasterrorid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(5, "done.\n");

    return err;
}

const char *
hipGetErrorString(hipError_t error)
{
    dscudaGetErrorStringResult *rp;

    initClient();
    WARN(5, "hipGetErrorString()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudageterrorstringid_1(error, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
    }
    WARN(5, "done.\n");

    return rp->errmsg;
}

/*
 * Device Management
 */

hipError_t
hipSetDeviceFlags(unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipSetDeviceFlags()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudasetdeviceflagsid_1(flags, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    invalidateModuleCache();

    WARN(3, "done.\n");

    return err;
}


hipError_t
hipDriverGetVersion (int *driverVersion)
{
    hipError_t err = hipSuccess;
    dscudaDriverGetVersionResult *rp;

    initClient();
    WARN(3, "cudaDriverGetVersionCount(0x%08llx)...", (unsigned long)driverVersion);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudadrivergetversionid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *driverVersion = rp->ver;
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipRuntimeGetVersion(int *runtimeVersion)
{
    hipError_t err = hipSuccess;
    dscudaRuntimeGetVersionResult *rp;

    initClient();
    WARN(3, "hipRuntimeGetVersion(0x%08llx)...", (unsigned long)runtimeVersion);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaruntimegetversionid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *runtimeVersion = rp->ver;
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceSynchronize(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipDeviceSynchronize()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudadevicesynchronize_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipDeviceReset(void)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipDeviceReset()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudadevicereset_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");

    return err;
}

/*
 * Execution Control
 */

hipError_t
hipFuncSetCacheConfig(reinterpret_cast<const void*>(const char * func), enum hipFuncCache_t cacheConfig)
{
    hipError_t err = hipSuccess;
    WARN(1, "Current implementation of hipFuncSetCacheConfig() does nothing "
         "but returning hipSuccess.\n");
    err = hipSuccess;
    return err;
}

hipError_t
dscudaFuncGetAttributesWrapper(int *moduleid, struct hipFuncAttributes *attr, const char *func)
{
    hipError_t err = hipSuccess;
    dscudaFuncGetAttributesResult *rp;

    initClient();
    WARN(3, "dscudaFuncGetAttributesWrapper(%d, 0x%08llx, %s)...",
         moduleid, (unsigned long)attr, func);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudafuncgetattributesid_1(moduleid[i], (char*)func, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    attr->binaryVersion      = rp->attr.binaryVersion;
    attr->constSizeBytes     = rp->attr.constSizeBytes;
    attr->localSizeBytes     = rp->attr.localSizeBytes;
    attr->maxThreadsPerBlock = rp->attr.maxThreadsPerBlock;
    attr->numRegs            = rp->attr.numRegs;
    attr->ptxVersion         = rp->attr.ptxVersion;
    attr->sharedSizeBytes    = rp->attr.sharedSizeBytes;
    WARN(3, "done.\n");
    WARN(3, "  attr->binaryVersion: %d\n", attr->binaryVersion);
    WARN(3, "  attr->constSizeBytes: %d\n", attr->constSizeBytes);
    WARN(3, "  attr->localSizeBytes: %d\n", attr->localSizeBytes);
    WARN(3, "  attr->maxThreadsPerBlock: %d\n", attr->maxThreadsPerBlock);
    WARN(3, "  attr->numRegs: %d\n", attr->numRegs);
    WARN(3, "  attr->ptxVersion: %d\n", attr->ptxVersion);
    WARN(3, "  attr->sharedSizeBytes: %d\n", attr->sharedSizeBytes);

    return err;
}

/*
 * Memory Management
 */

hipError_t
hipMalloc(void **devAdrPtr, size_t size)
{
    hipError_t err = hipSuccess;
    dscudaMallocResult *rp;

    initClient();
    WARN(3, "hipMalloc(0x%08llx, %d)...", (unsigned long)devAdrPtr, size);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamallocid_1(size, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *devAdrPtr = (void*)rp->devAdr;
    WARN(3, "done. *devAdrPtr:0x%08llx\n", *devAdrPtr);

    return err;
}

hipError_t
hipFree(void *mem)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipFree(0x%08llx)...", (unsigned long)mem);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudafreeid_1((RCadr)mem, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
hipMemcpy(void *dst, const void *src, size_t count, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaMemcpyD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    Vdev_t *vdev;
    RCServer_t *sp;

    initClient();

    WARN(3, "hipMemcpy(0x%08lx, 0x%08lx, %d, %s)...",
            (unsigned long)dst, (unsigned long)src, count, dscudaMemcpyKindName(kind));

    switch (kind) {
      case hipMemcpyDeviceToHost:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2hrp = dscudamemcpyd2hid_1((RCadr)src, count, Clnt[Vdevid][sp->id]);
            checkResult(d2hrp, sp);
            err = (hipError_t)d2hrp->err;
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            }
            else if (bcmp(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            }
            else {
                WARN(3, "hipMemcpy() data copied from device%d matched with that from device0.\n", i);
            }
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = count;
        srcbuf.RCbuf_val = (char *)src;
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpyh2did_1((RCadr)dst, srcbuf, count, Clnt[Vdevid][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpyd2did_1((RCadr)dst, (RCadr)src, count, Clnt[Vdevid][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }

    WARN(3, "done.\n");

    return err;
}

hipError_t
hipGetDeviceProperties(struct hipDeviceProp_t *prop, int device)
{
    hipError_t err = hipSuccess;
    dscudaGetDevicePropertiesResult *rp;

    initClient();
    WARN(3, "hipGetDeviceProperties(0x%08lx, %d)...", (unsigned long)prop, device);
    Vdev_t *vdev = Vdev + device;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudagetdevicepropertiesid_1(device, Clnt[device][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    memcpy(prop, rp->prop.RCbuf_val, rp->prop.RCbuf_len);
    WARN(3, "done.\n");

    return err;
}

/*
 * launch a kernel function of id 'kid', defined in a module of id 'moduleid'.
 * 'kid' must be unique inside a single module.
 */
void
dscudaLaunchKernelWrapper(int *moduleid, int kid, char *kname,
                         RCdim3 gdim, RCdim3 bdim, RCsize smemsize, RCstream stream,
                         RCargs args)
{
    RCmappedMem *mem;
    RCstreamArray *st;

    st = RCstreamArrayQuery((hipStream_t)stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }

    mem = RCmappedMemListTop;
    while (mem) {
        hipMemcpy(mem->pDevice, mem->pHost, mem->size, hipMemcpyHostToDevice);
        mem = mem->next;
    }

    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        void *rp = dscudalaunchkernelid_1(moduleid[i], kid, kname,
                                         gdim, bdim, smemsize, (RCstream)st->s[i],
                                         args, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
    }

    mem = RCmappedMemListTop;
    while (mem) {
        hipMemcpy(mem->pHost, mem->pDevice, mem->size, hipMemcpyDeviceToHost);
        mem = mem->next;
    }
}

void
ibvDscudaLaunchKernelWrapper(int *moduleid, int kid, char *kname,
                            int *gdim, int *bdim, RCsize smemsize, RCstream stream,
                            int narg, IbvArg *arg)
{
    // a dummy func.
}

hipError_t
hipMallocArray(struct hipArray **array, const struct hipChannelFormatDesc *desc,
                size_t width, size_t height, unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaMallocArrayResult *rp;
    RCchanneldesc descbuf;
    hipArray *ca[RC_NREDUNDANCYMAX];

    initClient();
    WARN(3, "hipMallocArray(0x%08llx, 0x%08llx, %d, %d, 0x%08x)...",
         (unsigned long)array, desc, width, height, flags);


    descbuf.x = desc->x;
    descbuf.y = desc->y;
    descbuf.z = desc->z;
    descbuf.w = desc->w;
    descbuf.f = desc->f;

    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamallocarrayid_1(descbuf, width, height, flags, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ca[i] = (hipArray *)rp->array;
    }

    *array = ca[0];
    RCcuarrayArrayRegister(ca);
    WARN(3, "done. *array:0x%08llx\n", *array);

    return err;
}

hipError_t
hipFreeArray(struct hipArray *array)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCcuarrayArray *ca;

    initClient();
    WARN(3, "hipFreeArray(0x%08llx)...", (unsigned long)array);
    ca = RCcuarrayArrayQuery(array);
    if (!ca) {
        WARN(0, "invalid hipArray : 0x%08llx\n", array);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudafreearrayid_1((RCadr)ca->ap[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    RCcuarrayArrayUnregister(ca->ap[0]);
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipMemcpyToArray(struct hipArray *dst, size_t wOffset, size_t hOffset, const void *src,
                  size_t count, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCcuarrayArray *ca;
    Vdev_t *vdev;
    RCServer_t *sp;

    initClient();

    WARN(3, "hipMemcpyToArray(0x%08llx, %d, %d, 0x%08llx, %d, %s)...",
         (unsigned long)dst, wOffset, hOffset, (unsigned long)src, count, dscudaMemcpyKindName(kind));
    ca = RCcuarrayArrayQuery(dst);
    if (!ca) {
        WARN(0, "invalid hipArray : 0x%08llx\n", dst);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = count;
        srcbuf.RCbuf_val = (char *)src;

        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpytoarrayh2did_1((RCadr)ca->ap[i], wOffset, hOffset, srcbuf, count, Clnt[Vdevid][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpytoarrayd2did_1((RCadr)ca->ap[i], wOffset, hOffset, (RCadr)src, count, Clnt[Vdevid][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
dscudaMemcpyToSymbolWrapper(int *moduleid, const char *symbol, const void *src,
                           size_t count, size_t offset, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCbuf srcbuf;
    RCServer_t *sp;
    Vdev_t *vdev;

    initClient();

    WARN(3, "dscudaMemcpyToSymbolWrapper(%d, 0x%08llx, 0x%08llx, %d, %d, %s)"
         "symbol:%s  ...",
         moduleid, (unsigned long)symbol, (unsigned long)src,
         count, offset, dscudaMemcpyKindName(kind), symbol);

    switch (kind) {
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = count;
        srcbuf.RCbuf_val = (char *)src;
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            rp = dscudamemcpytosymbolh2did_1(moduleid[i], (char *)symbol, srcbuf, count, offset, Clnt[Vdevid][sp->id]);
            checkResult(rp, sp);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            rp = dscudamemcpytosymbold2did_1(moduleid[i], (char *)symbol, (RCadr)src, count, offset, Clnt[Vdevid][sp->id]);
            checkResult(rp, sp);
            if (rp->err != hipSuccess) {
                err = (hipError_t)rp->err;
            }
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
dscudaMemcpyFromSymbolWrapper(int *moduleid, void *dst, const char *symbol,
                             size_t count, size_t offset,
                             enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaMemcpyFromSymbolD2HResult *d2hrp;
    dscudaResult *d2drp;
    Vdev_t *vdev;
    RCServer_t *sp;

    initClient();

    WARN(3, "dscudaMemcpyFromSymbolWrapper(0x%08llx, 0x%08llx, 0x%08llx, %d, %d, %s)"
         "symbol:%s  ...",
         moduleid, (unsigned long)dst, (unsigned long)symbol,
         count, offset, dscudaMemcpyKindName(kind), symbol);

    switch (kind) {
      case hipMemcpyDeviceToDevice:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpyfromsymbold2did_1(moduleid[i], (RCadr)dst, (char *)symbol, count, offset, Clnt[Vdevid][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
        }
        break;
      case hipMemcpyDeviceToHost:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2hrp = dscudamemcpyfromsymbold2hid_1(moduleid[i], (char *)symbol, count, offset, Clnt[Vdevid][sp->id]);
            checkResult(d2hrp, sp);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            }
            else if (bcmp(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            }
            else {
                WARN(3, "hipMemcpyFromSymbol() data copied from device%d matched with that from device0.\n", i);
            }
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemset(void *devPtr, int value, size_t count)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipMemset()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamemsetid_1((RCadr)devPtr, value, count, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMallocPitch(void **devPtr, size_t *pitch, size_t width, size_t height)
{
    hipError_t err = hipSuccess;
    dscudaMallocPitchResult *rp;

    initClient();
    WARN(3, "hipMallocPitch(0x%08llx, 0x%08llx, %d, %d)...",
         (unsigned long)devPtr, pitch, width, height);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamallocpitchid_1(width, height, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *devPtr = (void*)rp->devPtr;
    *pitch = rp->pitch;
    WARN(3, "done. *devPtr:0x%08llx  *pitch:%d\n", *devPtr, *pitch);

    return err;
}

hipError_t
hipMemcpy2DToArray(struct hipArray *dst, size_t wOffset, size_t hOffset,
                    const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaMemcpy2DToArrayD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCcuarrayArray *ca;
    Vdev_t *vdev;
    RCServer_t *sp;

    initClient();

    WARN(3, "hipMemcpy2DToArray(0x%08llx, %d, %d, 0x%08llx, %d, %d, %d, %s)...",
         (unsigned long)dst, wOffset, hOffset,
         (unsigned long)src, spitch, width, height, dscudaMemcpyKindName(kind));
    ca = RCcuarrayArrayQuery(dst);
    if (!ca) {
        WARN(0, "invalid hipArray : 0x%08llx\n", dst);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyDeviceToHost:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2hrp = dscudamemcpy2dtoarrayd2hid_1(wOffset, hOffset,
                                                (RCadr)src, spitch, width, height, Clnt[Vdevid][sp->id]);
            checkResult(d2hrp, sp);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            }
            else if (bcmp(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            }
            else {
                WARN(3, "hipMemcpy2DToArray() data copied from device%d matched with that from device0.\n", i);
            }
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = spitch * height;
        srcbuf.RCbuf_val = (char *)src;
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpy2dtoarrayh2did_1((RCadr)ca->ap[i], wOffset, hOffset,
                                                srcbuf, spitch, width, height, Clnt[Vdevid][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpy2dtoarrayd2did_1((RCadr)ca->ap[i], wOffset, hOffset,
                                                (RCadr)src, spitch, width, height, Clnt[Vdevid][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemcpy2D(void *dst, size_t dpitch,
             const void *src, size_t spitch, size_t width, size_t height, enum hipMemcpyKind kind)
{
    hipError_t err = hipSuccess;
    dscudaMemcpy2DD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    Vdev_t *vdev;
    RCServer_t *sp;

    initClient();

    WARN(3, "hipMemcpy2D(0x%08llx, %d, 0x%08llx, %d, %d, %d, %s)...",
         (unsigned long)dst, dpitch,
         (unsigned long)src, spitch, width, height, dscudaMemcpyKindName(kind));

    switch (kind) {
      case hipMemcpyDeviceToHost:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2hrp = dscudamemcpy2dd2hid_1(dpitch,
                                         (RCadr)src, spitch, width, height, Clnt[Vdevid][sp->id]);
            checkResult(d2hrp, sp);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
            if (i == 0) {
                memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
            }
            else if (bcmp(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len) != 0) {
                if (errorHandler) {
                    errorHandler(errorHandlerArg);
                }
            }
            else {
                WARN(3, "hipMemcpy() data copied from device%d matched with that from device0.\n", i);
            }
        }
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = spitch * height;
        srcbuf.RCbuf_val = (char *)src;
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpy2dh2did_1((RCadr)dst, dpitch,
                                         srcbuf, spitch, width, height, Clnt[Vdevid][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
        }
        break;
      case hipMemcpyDeviceToDevice:
        vdev = Vdev + Vdevid;
        sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpy2dd2did_1((RCadr)dst, dpitch,
                                         (RCadr)src, spitch, width, height, Clnt[Vdevid][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipMemset2D(void *devPtr, size_t pitch, int value, size_t width, size_t height)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipMemset2D(0x%08llx, %d, %d, %d, %d)...",
         (unsigned long)devPtr, pitch, value, width, height);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamemset2did_1((RCadr)devPtr, pitch, value, width, height, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");

    return err;
}

hipError_t
hipHostMalloc(void **ptr, size_t size)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaMallocHostResult *rp;

    initClient();
    WARN(3, "hipHostMalloc(0x%08llx, %d)...", (unsigned long)ptr, size);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudamallochostid_1(size, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *ptr = (void*)rp->ptr;

    WARN(3, "done. *ptr:0x%08llx\n", *ptr);
    return err;
#else
    // returned memory is not page locked.
    // it cannot be passed to hipMemcpyAsync().
    *ptr = malloc(size);
    if (*ptr) {
        return hipSuccess;
    }
    else {
        return hipErrorOutOfMemory;
    }
#endif
}

hipError_t
hipHostAlloc(void **pHost, size_t size, unsigned int flags)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaHostAllocResult *rp;

    initClient();
    WARN(3, "hipHostAlloc(0x%08llx, %d, 0x%08x)...", (unsigned long)pHost, size, flags);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudahostallocid_1(size, flags, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *pHost = (void*)rp->pHost;
    WARN(3, "done. *pHost:0x%08llx\n", *pHost);

    return err;
#else
    // returned memory is not page locked.
    // it cannot be passed to hipMemcpyAsync().

    hipError_t err = hipSuccess;
    void *devmem;

    initClient();
    WARN(3, "hipHostAlloc(0x%08llx, %d, 0x%08x)...", (unsigned long)pHost, size, flags);

    *pHost = malloc(size);
    if (!*pHost) return hipErrorOutOfMemory;
    if (!(flags & hipHostMallocMapped)) {
        WARN(3, "done. *pHost:0x%08llx\n", *pHost);
        return hipSuccess;
    }

    // flags says the host memory must be mapped on to the device memory.
    err = hipMalloc(&devmem, size);
    if (err == hipSuccess) {
        RCmappedMemRegister(*pHost, devmem, size);
    }
    WARN(3, "done. host mem:0x%08llx  device mem:0x%08llx\n", *pHost, devmem);

    return err;
#endif
}

hipError_t
hipHostFree(void *ptr)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaResult *rp;

    initClient();
    WARN(3, "hipHostFree(0x%08llx)...", (unsigned long)ptr);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudafreehostid_1((RCadr)ptr, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");
    return err;
#else
    hipError_t err = hipSuccess;
    RCmappedMem *mem = RCmappedMemQuery(ptr);
    free(ptr);
    if (mem) { // ptr mapped on to a device memory.
        err = hipFree(mem->pDevice);
        RCmappedMemUnregister(ptr);
        return err;
    }
    else {
        return hipSuccess;
    }
#endif
}

// flags is not used for now in CUDA3.2. It should always be zero.
hipError_t
hipHostGetDevicePointer(void **pDevice, void*pHost, unsigned int flags)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaHostGetDevicePointerResult *rp;

    initClient();
    WARN(3, "hipHostGetDevicePointer(0x%08llx, 0x%08llx, 0x%08x)...",
         (unsigned long)pDevice, (unsigned long)pHost, flags);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudahostgetdevicepointerid_1((RCadr)pHost, flags, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *pDevice = (void *)rp->pDevice;
    WARN(3, "done. *pDevice:0x%08llx\n", *pDevice);
    return err;
#else
    RCmappedMem *mem = RCmappedMemQuery(pHost);
    if (!mem) return hipErrorInvalidValue; // pHost is not registered as RCmappedMem.
    *pDevice = mem->pDevice;
    return hipSuccess;
#endif
}

hipError_t
hipHostGetFlags(unsigned int *pFlags, void *pHost)
{
    hipError_t err = hipSuccess;
    dscudaHostGetFlagsResult *rp;

    initClient();
    WARN(3, "hipHostGetFlags(0x%08x 0x%08llx)...",
         (unsigned long)pFlags, (unsigned long)pHost);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudahostgetflagsid_1((RCadr)pHost, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    *pFlags = rp->flags;
    WARN(3, "done. flags:0x%08x\n", *pFlags);
    return err;
    
}

hipError_t
hipMemcpyAsync(void *dst, const void *src, size_t count, enum hipMemcpyKind kind, hipStream_t stream)
{
#if RC_SUPPORT_PAGELOCK
    hipError_t err = hipSuccess;
    dscudaMemcpyAsyncD2HResult *d2hrp;
    dscudaResult *h2drp, *d2drp;
    RCbuf srcbuf;
    RCstreamArray *st;
    Vdev_t *vdev;
    RCServer_t *sp;

    initClient();

    WARN(3, "hipMemcpyAsync(0x%08llx, 0x%08llx, %d, %s, 0x%08llx)...",
         (unsigned long)dst, (unsigned long)src, count, dscudaMemcpyKindName(kind), st->s[0]);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    switch (kind) {
      case hipMemcpyDeviceToHost:
        Vdev_t *vdev = Vdev + Vdevid;
        RCServer_t *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2hrp = dscudamemcpyasyncd2hid_1((RCadr)src, count, (RCstream)st->s[i], Clnt[Vdevid][sp->id]);
            checkResult(d2hrp, sp);
            if (d2hrp->err != hipSuccess) {
                err = (hipError_t)d2hrp->err;
            }
        }
        memcpy(dst, d2hrp->buf.RCbuf_val, d2hrp->buf.RCbuf_len);
        break;
      case hipMemcpyHostToDevice:
        srcbuf.RCbuf_len = count;
        srcbuf.RCbuf_val = (char *)src;
        Vdev_t *vdev = Vdev + Vdevid;
        RCServer_t *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            h2drp = dscudamemcpyasynch2did_1((RCadr)dst, srcbuf, count, (RCstream)st->s[i], Clnt[Vdevid][sp->id]);
            checkResult(h2drp, sp);
            if (h2drp->err != hipSuccess) {
                err = (hipError_t)h2drp->err;
            }
        }
        break;
      case hipMemcpyDeviceToDevice:
        Vdev_t *vdev = Vdev + Vdevid;
        RCServer_t *sp = vdev->server;
        for (int i = 0; i < vdev->nredundancy; i++, sp++) {
            d2drp = dscudamemcpyasyncd2did_1((RCadr)dst, (RCadr)src, count, (RCstream)st->s[i], Clnt[Vdevid][sp->id]);
            checkResult(d2drp, sp);
            if (d2drp->err != hipSuccess) {
                err = (hipError_t)d2drp->err;
            }
        }
        break;
      default:
        WARN(0, "Unsupported value for hipMemcpyKind : %s\n", dscudaMemcpyKindName(kind));
        exit(1);
    }
    WARN(3, "done.\n");

    return err;

#else
    // this DOES block.
    // this is only for use with a poor implementation of dscudaMallocHost().
    return hipMemcpy(dst, src, count, kind);
#endif
}

/*
 * Stream Management
 */

hipError_t
hipStreamCreate(hipStream_t *pStream)
{
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaStreamCreateResult *rp;
    hipStream_t st[RC_NREDUNDANCYMAX];

    initClient();
    WARN(3, "hipStreamCreate(0x%08llx)...", (unsigned long)pStream);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamcreateid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        st[i] = (hipStream_t)rp->stream;
    }

    *pStream = st[0];
    RCstreamArrayRegister(st);
    WARN(3, "done. *pStream:0x%08llx\n", *pStream);

    return err;
#else
    *pStream = 0;
    return hipSuccess;
#endif
}

hipError_t
hipStreamDestroy(hipStream_t stream)
{
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    initClient();
    WARN(3, "hipStreamDestroy(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamdestroyid_1((RCadr)st->s[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    RCstreamArrayUnregister(st->s[0]);
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

hipError_t
hipStreamSynchronize(hipStream_t stream)
{
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    initClient();
    WARN(3, "hipStreamSynchronize(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamsynchronizeid_1((RCadr)st->s[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

hipError_t
hipStreamQuery(hipStream_t stream)
{
#if RC_SUPPORT_STREAM
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;

    initClient();
    WARN(3, "hipStreamQuery(0x%08llx)...", (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamqueryid_1((RCadr)st->s[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");
    return err;
#else
    return hipSuccess;
#endif
}

/*
 * Event Management
 */

hipError_t
hipEventCreate(hipEvent_t *event)
{
    hipError_t err = hipSuccess;
    dscudaEventCreateResult *rp;
    hipEvent_t ev[RC_NREDUNDANCYMAX];

    initClient();
    WARN(3, "hipEventCreate(0x%08llx)...", (unsigned long)event);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventcreateid_1(Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ev[i] = (hipEvent_t)rp->event;
    }
    *event = ev[0];
    RCeventArrayRegister(ev);
    WARN(3, "done. *event:0x%08llx\n", *event);

    return err;
}

hipError_t
hipEventCreateWithFlags(hipEvent_t *event, unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaEventCreateResult *rp;
    hipEvent_t ev[RC_NREDUNDANCYMAX];

    initClient();
    WARN(3, "hipEventCreateWithFlags(0x%08llx, 0x%08x)...", (unsigned long)event, flags);
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventcreatewithflagsid_1(flags, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
        ev[i] = (hipEvent_t)rp->event;
    }
    *event = ev[0];
    RCeventArrayRegister(ev);
    WARN(3, "done. *event:0x%08llx\n", *event);

    return err;
}

hipError_t
hipEventDestroy(hipEvent_t event)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    initClient();
    WARN(3, "hipEventDestroy(0x%08llx)...", (unsigned long)event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : 0x%08llx\n", event);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventdestroyid_1((RCadr)ev->e[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    RCeventArrayUnregister(ev->e[0]);
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventElapsedTime(float *ms, hipEvent_t start, hipEvent_t end)
{
    hipError_t err = hipSuccess;
    dscudaEventElapsedTimeResult *rp;
    RCeventArray *es, *ee;

    initClient();
    WARN(3, "hipEventElapsedTime(0x%08llx, 0x%08llx, 0x%08llx)...",
         (unsigned long)ms, (unsigned long)start, (unsigned long)end);
    es = RCeventArrayQuery(start);
    if (!es) {
        WARN(0, "invalid start event : 0x%08llx\n", start);
        exit(1);
    }
    ee = RCeventArrayQuery(end);
    if (!ee) {
        WARN(0, "invalid end event : 0x%08llx\n", end);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventelapsedtimeid_1((RCadr)es->e[i], (RCadr)ee->e[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    *ms = rp->ms;
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventRecord(hipEvent_t event, hipStream_t stream)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;
    RCeventArray *ev;

    initClient();
    WARN(3, "hipEventRecord(0x%08llx, 0x%08llx)...", (unsigned long)event, (unsigned long)stream);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : 0x%08llx\n", event);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventrecordid_1((RCadr)ev->e[i], (RCadr)st->s[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventSynchronize(hipEvent_t event)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    initClient();
    WARN(3, "hipEventSynchronize(0x%08llx)...", (unsigned long)event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : 0x%08llx\n", event);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventsynchronizeid_1((RCadr)ev->e[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipEventQuery(hipEvent_t event)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCeventArray *ev;

    initClient();
    WARN(3, "hipEventQuery(0x%08llx)...", (unsigned long)event);
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : 0x%08llx\n", event);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudaeventqueryid_1((RCadr)ev->e[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
hipStreamWaitEvent(hipStream_t stream, hipEvent_t event, unsigned int flags)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCstreamArray *st;
    RCeventArray *ev;

    initClient();
    WARN(3, "hipStreamWaitEvent(0x%08llx, 0x%08llx, 0x%08x)...",
         (unsigned long)stream, (unsigned long)event, flags);
    st = RCstreamArrayQuery(stream);
    if (!st) {
        WARN(0, "invalid stream : 0x%08llx\n", stream);
        exit(1);
    }
    ev = RCeventArrayQuery(event);
    if (!ev) {
        WARN(0, "invalid event : 0x%08llx\n", event);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudastreamwaiteventid_1((RCadr)st->s[i], (RCadr)ev->e[i], flags, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    WARN(3, "done.\n");
    return err;
}

/*
 * Texture Reference Management
 */

hipChannelFormatDesc
hipCreateChannelDesc(int x, int y, int z, int w, enum hipChannelFormatKind f)
{
    dscudaCreateChannelDescResult *rp;
    hipChannelFormatDesc desc;

    initClient();
    WARN(3, "hipCreateChannelDesc()...");
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudacreatechanneldescid_1(x, y, z, w, f, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
    }
    WARN(3, "done.\n");

    desc.x = rp->x;
    desc.y = rp->y;
    desc.z = rp->z;
    desc.w = rp->w;
    desc.f = (enum hipChannelFormatKind)rp->f;

    return desc;
}

hipError_t
hipGetChannelDesc(struct hipChannelFormatDesc *desc, const struct hipArray *array)
{
    hipError_t err = hipSuccess;
    dscudaGetChannelDescResult *rp;
    RCcuarrayArray *ca;

    initClient();
    WARN(3, "hipGetChannelDesc()...");
    ca = RCcuarrayArrayQuery((hipArray *)array);
    if (!ca) {
        WARN(0, "invalid hipArray : 0x%08llx\n", array);
        exit(1);
    }
    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudagetchanneldescid_1((RCadr)ca->ap[i], Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    WARN(3, "done.\n");

    desc->x = rp->x;
    desc->y = rp->y;
    desc->z = rp->z;
    desc->w = rp->w;
    desc->f = (enum hipChannelFormatKind)rp->f;

    WARN(3, "done.\n");
    return err;
}

hipError_t
dscudaBindTextureWrapper(int *moduleid, char *texname,
                        size_t *offset,
                        const struct textureReference *tex,
                        const void *devPtr,
                        const struct hipChannelFormatDesc *desc,
                        size_t size)
{
    hipError_t err = hipSuccess;
    dscudaBindTextureResult *rp;
    RCtexture texbuf;

    initClient();

    WARN(3, "dscudaBindTextureWrapper(0x%08llx, %s, 0x%08llx, 0x%08llx, 0x%08llx, 0x%08llx, %d)...",
         moduleid, texname,
         offset, tex, devPtr, desc, size);

    setTextureParams(&texbuf, tex, desc);

    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudabindtextureid_1(moduleid[i], texname,
                                  (RCadr)devPtr, size, (RCtexture)texbuf, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    if (offset) {
        *offset = rp->offset;
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
dscudaBindTexture2DWrapper(int *moduleid, char *texname,
                          size_t *offset,
                          const struct textureReference *tex,
                          const void *devPtr,
                          const struct hipChannelFormatDesc *desc,
                          size_t width, size_t height, size_t pitch)
{
    hipError_t err = hipSuccess;
    dscudaBindTexture2DResult *rp;
    RCtexture texbuf;

    initClient();

    WARN(3, "dscudaBindTexture2DWrapper(0x%08llx, %s, 0x%08llx, 0x%08llx, 0x%08llx, 0x%08llx, %d, %d, %d)...",
         moduleid, texname,
         offset, tex, devPtr, desc, width, height, pitch);

    setTextureParams(&texbuf, tex, desc);

    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudabindtexture2did_1(moduleid[i], texname,
                                    (RCadr)devPtr, width, height, pitch, (RCtexture)texbuf, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }

    if (offset) {
        *offset = rp->offset;
    }

    WARN(3, "done.\n");
    return err;
}

hipError_t
dscudaBindTextureToArrayWrapper(int *moduleid, char *texname,
                               const struct textureReference *tex,
                               const struct hipArray *array,
                               const struct hipChannelFormatDesc *desc)
{
    hipError_t err = hipSuccess;
    dscudaResult *rp;
    RCtexture texbuf;
    RCcuarrayArray *ca;

    initClient();

    WARN(3, "dscudaBindTextureToArrayWrapper(0x%08llx, %s, 0x%08llx, 0x%08llx)...",
         moduleid, texname, (unsigned long)array, (unsigned long)desc);

    setTextureParams(&texbuf, tex, desc);

    ca = RCcuarrayArrayQuery((hipArray *)array);
    if (!ca) {
        WARN(0, "invalid hipArray : 0x%08llx\n", array);
        exit(1);
    }

    Vdev_t *vdev = Vdev + Vdevid;
    RCServer_t *sp = vdev->server;
    for (int i = 0; i < vdev->nredundancy; i++, sp++) {
        rp = dscudabindtexturetoarrayid_1(moduleid[i], texname, (RCadr)ca->ap[i], (RCtexture)texbuf, Clnt[Vdevid][sp->id]);
        checkResult(rp, sp);
        if (rp->err != hipSuccess) {
            err = (hipError_t)rp->err;
        }
    }
    WARN(3, "done.\n");
    return err;
}

hipError_t
hipUnbindTexture(const struct textureReference * texref)
{
    hipError_t err = hipSuccess;

    WARN(4, "Current implementation of hipUnbindTexture() does nothing "
         "but returning hipSuccess.\n");

    err = hipSuccess;

    return err;
}

#include "libdscuda.cu"
