#include <string.h>
#include "dscuda.h"
#include "dscudarpc.h"
#include "dscudaverb.h"

static verbHist *verbHists = NULL;
static int verbHistNum = 0;
static int verbHistMax = 0;

void verbAddHist(int funcID, cudaArgs args) {
    if (verbHistNum == verbHistMax) {
        verbHistMax += 10;
        verbHists = (verbHist *)realloc(verbHists, sizeof(verbHist) * verbHistMax);
    }

    if (funcID == dscudaLaunchKernelId) {
        if (dscudaRemoteCallType() == RC_REMOTECALL_TYPE_IBV) { // IBV
#warning fix this part
        }
        else { // RPC
            RCargs *LKargs = &(args.rpcCudaLaunchKernelArgs.args);
            RCarg *args2;

            args2 = (RCarg *)malloc(sizeof(RCarg) * LKargs->RCargs_len);
            memcpy(args2, LKargs->RCargs_val, sizeof(RCarg) * LKargs->RCargs_len);
            LKargs->RCargs_val = args2;
        }
    }

    verbHists[verbHistNum].funcID = funcID;
    verbHists[verbHistNum].args = args;
    verbHistNum++;

    //WARN(3, "%dth function history added\n", verbHistNum);
    return;
}

void verbClearHist(void) {
    verbHistNum = verbHistMax = 0;
    if (verbHists) {
        free(verbHists);
        verbHists = NULL;
    }

    WARN(3, "function history cleared\n");
    return;
}

void verbRecallHist(void) {
    int i;
    int *moduleid;

    dscudaSetAutoVerb(0);
    WARN(1, "\nIllegal return value has detected\nRecalling functions...\n");

    for (i = 0; i < verbHistNum; i++) {
        switch (verbHists[i].funcID) {
            {
              case dscudaSetDeviceId:
                WARN(3, "recall hipSetDevice\n");
                cudaSetDeviceArgsType args = verbHists[i].args.cudaSetDeviceArgs;
                hipSetDevice(args.device);
                break;
            }

            {
              case dscudaGetDevicePropertiesId:
                WARN(3, "recall hipGetDeviceProperties [not implemented yet]\n");
                cudaGetDevicePropertiesArgsType args = verbHists[i].args.cudaGetDevicePropertiesArgs;
                //hipGetDeviceProperties(args.prop, args.device);
                break;
            }

            {
              case dscudaMallocId:
                WARN(3, "recall hipMalloc\n");
                cudaMallocArgsType args = verbHists[i].args.cudaMallocArgs;
                hipMalloc(args.devPtr, args.size);
                break;
            }

            {
              case dscudaMemcpyH2DId:
                WARN(3, "recall hipMemcpy H2D\n");
                cudaMemcpyArgsType args = verbHists[i].args.cudaMemcpyArgs;
                hipMemcpy(args.dst, args.src, args.count, args.kind);
                break;
            }

            {
              case dscudaMemcpyD2DId:
                WARN(3, "recall hipMemcpy D2D\n");
                cudaMemcpyArgsType args = verbHists[i].args.cudaMemcpyArgs;
                hipMemcpy(args.dst, args.src, args.count, args.kind);
                break;
            }

            {
              case dscudaMemcpyD2HId:
                WARN(3, "recall hipMemcpy D2H\n");
                cudaMemcpyArgsType args = verbHists[i].args.cudaMemcpyArgs;
                hipMemcpy(args.dst, args.src, args.count, args.kind);
                break;
            }

            {
              case dscudaMemcpyToSymbolH2DId:
                WARN(3, "recall MemcpyToSymbol\n");
                cudaMemcpyToSymbolArgsType args = verbHists[i].args.cudaMemcpyToSymbolArgs;
                dscudaMemcpyToSymbolWrapper(moduleid, args.symbol, args.src, args.count, args.offset, args.kind);
                break;
            }

            {
              case dscudaFreeId:
                WARN(3, "recall hipFree\n");
                cudaFreeArgsType args = verbHists[i].args.cudaFreeArgs;
                hipFree(args.devPtr);
                break;
            }

            {
              case dscudaLoadModuleId:
                WARN(3, "recall cudaLoadModule\n");
                cudaLoadModuleArgsType args = verbHists[i].args.cudaLoadModuleArgs;
                moduleid = dscudaLoadModule(args.srcname);
                break;
            }

            {
              case dscudaLaunchKernelId:
                WARN(3, "recall hipLaunchKernel\n");
                if (dscudaRemoteCallType() == RC_REMOTECALL_TYPE_IBV) {
                    ibvCudaLaunchKernelArgsType args = verbHists[i].args.ibvCudaLaunchKernelArgs;
                    ibvDscudaLaunchKernelWrapper(args.moduleid, args.kid, args.kname,
                                                 args.gdim, args.bdim, args.smemsize, args.stream,
                                                 args.narg, args.arg);
                    free(args.arg);
                }
                else {
                    rpcCudaLaunchKernelArgsType args = verbHists[i].args.rpcCudaLaunchKernelArgs;
                    rpcDscudaLaunchKernelWrapper(args.moduleid, args.kid, args.kname,
                                                 args.gdim, args.bdim, args.smemsize, args.stream,
                                                 args.args);
                    free(args.args.RCargs_val);
                }
                break;
            }
        }
    }

    dscudaSetAutoVerb(1);

    WARN(1, "done.\n");
}
