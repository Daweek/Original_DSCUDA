
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <netdb.h>
#include <arpa/inet.h>
#include <sys/time.h>
#include <hip/driver_types.h>

static int WarnLevel = 2; /* warning message output level. the higher the more verbose.
                             0: no warning (may cause wrong result with g7pkg/scripts/check.csh)
                             1: minimum
                             2: default
                             3: for debugging purpose
                          */
int
dscudaWarnLevel(void)
{
    return WarnLevel;
}

void
dscudaSetWarnLevel(int level)
{
    WarnLevel = level;
}

char *
dscudaMemcpyKindName(hipMemcpyKind kind)
{
    static char *name;

    switch (kind) {
      case hipMemcpyHostToHost:
        name = "hipMemcpyHostToHost";
        break;
      case hipMemcpyHostToDevice:
        name = "hipMemcpyHostToDevice";
        break;
      case hipMemcpyDeviceToHost:
        name = "hipMemcpyDeviceToHost";
        break;
      case hipMemcpyDeviceToDevice:
        name = "hipMemcpyDeviceToDevice";
        break;
      case hipMemcpyDefault:
        name = "hipMemcpyDefault";
        break;
      default:
        name = "Invalid hipMemcpyKind";
    }
    return name;
}

unsigned int
dscudaServerNameToDevid(char *svrname)
{
    char buf[256];
    char *token, *savep = NULL;

    strncpy(buf, svrname, sizeof(buf));
    token = strtok_r(buf, ":", &savep);
    token = strtok_r(NULL, ":", &savep);
    return token ? atoi(token) : 0;
}

unsigned int
dscudaServerIpStrToAddr(char *ipstr)
{
    unsigned int addr;
    struct hostent *host;
    host = gethostbyname(ipstr);
    if (!host) {
        fprintf(stderr, "unknown host name: %s\n", ipstr);
        exit(1);
    }
    addr = *(unsigned int *)host->h_addr_list[0];

    return addr;
}

unsigned int
dscudaServerNameToAddr(char *svrname)
{
    char buf[256];
    char *token, *savep = NULL;

    strncpy(buf, svrname, sizeof(buf));
    token = strtok_r(buf, ":", &savep);
    return dscudaServerIpStrToAddr(token);
}

char *
dscudaAddrToServerIpStr(unsigned int addr)
{
    struct in_addr ia;

    ia.s_addr = addr;
    return inet_ntoa(ia);
}

/*
 *
 * t0 : time of day (in second) the last time this function is called.
 * returns the number of seconds passed since *t0.
 */
double
RCgetCputime(double *t0)
{
    struct timeval t;
    double tnow, dt;

    gettimeofday(&t, NULL);
    tnow = t.tv_sec + t.tv_usec/1000000.0;
    dt = tnow - *t0;
    *t0 = tnow;
    return dt;
}


int
dscudaAlignUp(int off, int align)
{
    return ((off) + (align) - 1) & ~((align) - 1);
}

unsigned int
dscudaRoundUp(unsigned int src, unsigned int by)
{
    unsigned int dst = ((src - 1) / by + 1) * by;
    return dst;
}

